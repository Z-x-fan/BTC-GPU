#include "hip/hip_runtime.h"
#include <>
#include <hip/hip_runtime.h>
#include<iostream>
#include <vector>
#include<Windows.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include<queue>
#include <bitset>
#include<list>
#include<ctime>
#include <random>
#include <fstream>
#include<sstream>
#include "tensor_SGD.h"
#include "sgd_Kernel.h"
#include "block.h"

using namespace std;

double get_time_2(void)
{
	LARGE_INTEGER timer;
	static LARGE_INTEGER fre;
	static int init = 0;
	double t;

	if (init != 1) {
		QueryPerformanceFrequency(&fre);
		init = 1;
	}

	QueryPerformanceCounter(&timer);

	t = timer.QuadPart * 1. / (double)fre.QuadPart;

	return t;
}
double RMSE_GPU_S(double* s, double* a, double* b, double* c) {
	double total_error = 0;
	int num = 0;
	for (int i = 0; i < I; i++) {
		for (int j = 0; j < J; j++) {
			for (int m = 0; m < K; m++) {

				if (s[i * J * K + j * K + m] > 0) {
					num++;
					double dot = 0;
					for (int n = 0; n < r; n++) {
						dot += a[i * r + n] * b[j * r + n] * c[m * r + n];
					}
					double error = s[i * J * K + j * K + m] - dot;
					total_error += pow(error, 2);
				}
			}

		}
	}
	double Error = sqrt(total_error / num);
	return Error;
}
double RMSE_GPU_T(double* s, double* t, double* a, double* b, double* c) {
	double total_error = 0;
	int num = 0;
	for (int i = 0; i < I; i++) {
		for (int j = 0; j < J; j++) {
			for (int m = 0; m < K; m++) {

				if (s[i * J * K + j * K + m] == 0) {
					num++;
					double dot = 0;
					for (int n = 0; n < r; n++) {
						dot += a[i * r + n] * b[j * r + n] * c[m * r + n];
					}
					double error = t[i * J * K + j * K + m] - dot;
					total_error += pow(error, 2);
				}
			}

		}
	}
	double Error = sqrt(total_error / num);
	return Error;
}


__global__ void tensor_GPU(double* dX, b_node* d_bs, int* d_num_bs, double* dA, double* dB, double* dC, int num_parallel, int I, int J, int K, int r, double lr, double reg, Node_conflict* B_conf) {
	int bx = blockIdx.x;
	int len = 0;
	//printf("%d %d %d   ", bx, num_parallel, d_num_bs[0]);
	for (int i = 0; i < num_parallel; i++) {
		
		//printf("%d %d %d %d   ", bx, num_parallel, i, d_num_bs[0]);
		if (bx < d_num_bs[i]) {
			//printf("%d %d %d %d   ", bx, num_parallel, d_num_bs[i], d_bs[0].x_start);
			int x_start = d_bs[bx + len].x_start;
			int x_end = d_bs[bx + len].x_end;
			int x_s = x_end - x_start;
			int y_start = d_bs[bx + len].y_start;
			int y_end = d_bs[bx + len].y_end;
			int y_s = y_end - y_start;
			int z_start = d_bs[bx + len].z_start;
			int z_end = d_bs[bx + len].z_end;
			int z_s = z_end - z_start;
			int id = d_bs[bx + len].id;
			int cox = B_conf[id].coe_x;
			int coy = B_conf[id].coe_y;
			int coz = B_conf[id].coe_z;
			int tx = threadIdx.x;
			int ty = threadIdx.x;
			int tz = threadIdx.x;
			//printf("%d %d %d    ",bx, d_num_bs[1], d_bs[bx + len].x_start);
			if (tx < x_s && ty < y_s && tz < z_s && tx == ty && ty == tz) {
				int min = 0;
				min = x_s < y_s ? x_s : y_s;
				min = min < z_s ? min : z_s;
				int x = tx + x_start;
				int y = tx + y_start;
				int z = tz + z_start;
				//printf("%d %d %d %d %d %d %d %d %d\n", d_num_bs[i], bx, i, x_start, x_end, y_start, y_end, z_start, z_end);
				if (min == x_s) {
					for (int count = 0; count < y_s * z_s; count++) {

						if (dX[x * J * K + y * K + z] > 0) {
							//printf("%d %d %d %f\n", x, y, z, dX[x * J * K + y * K + z]);
							double dot = 0;
							for (int n = 0; n < r; n++) {
								dot += dA[x * r + n] * dB[y * r + n] * dC[z * r + n];
							}
							double error = dX[x * J * K + y * K + z] - dot;
							

							for (int n = 0; n < r; n++) {
								dA[x * r + n] += cox * lr * (error * dB[y * r + n] * dC[z * r + n] - reg * dA[x * r + n]);
								dB[y * r + n] += coy * lr * (error * dA[x * r + n] * dC[z * r + n] - reg * dB[y * r + n]);
								dC[z * r + n] += coz * lr * (error * dB[y * r + n] * dA[x * r + n] - reg * dC[z * r + n]);
							}
						}
						if (z == z_end - 1) {
							y = (y + 1) % y_end;
						}
						z = (z + 1) % z_end;

					}
				}

				if (min == y_s) {

					for (int count = 0; count < x_s * z_s; count++) {

						if (dX[x * J * K + y * K + z] > 0) {
							double dot = 0;
							for (int n = 0; n < r; n++) {
								dot += dA[x * r + n] * dB[y * r + n] * dC[z * r + n];
							}
							double error = dX[x * J * K + y * K + z] - dot;

							for (int n = 0; n < r; n++) {
								dA[x * r + n] += lr * (error * dB[y * r + n] * dC[z * r + n] - reg * dA[x * r + n]);
								dB[y * r + n] += lr * (error * dA[x * r + n] * dC[z * r + n] - reg * dB[y * r + n]);
								dC[z * r + n] += lr * (error * dB[y * r + n] * dA[x * r + n] - reg * dC[z * r + n]);
							}
						}

						if (z == x_end - 1) {
							x = (x + 1) % x_end;
						}
						z = (z + 1) % z_end;
					}

				}

				if (min == z_s) {

					for (int count = 0; count < x_s * y_s; count++) {

						if (dX[x * J * K + y * K + z] > 0) {
							double dot = 0;
							for (int n = 0; n < r; n++) {
								dot += dA[x * r + n] * dB[y * r + n] * dC[z * r + n];
							}
							double error = dX[x * J * K + y * K + z] - dot;

							for (int n = 0; n < r; n++) {
								dA[x * r + n] += lr * (error * dB[y * r + n] * dC[z * r + n] - reg * dA[x * r + n]);
								dB[y * r + n] += lr * (error * dA[x * r + n] * dC[z * r + n] - reg * dB[y * r + n]);
								dC[z * r + n] += lr * (error * dB[y * r + n] * dA[x * r + n] - reg * dC[z * r + n]);
							}
						}

						if (y == z_end - 1) {
							x = (x + 1) % x_end;
						}
						y = (y + 1) % y_end;
					}

				}
			}

		}
		len = len + d_num_bs[i];
	}

}

//lack_free
__global__ void tensor_LF(LF_node* d_LF, int* d_num_LF, double* dA, double* dB, double* dC, int num_parallel, int thread_size, int I, int J, int K, int r, double lr, double reg) {
	int len = 0;
	for (int i = 0; i < num_parallel; i++) {
		int tx = threadIdx.x;
		if (thread_size < d_num_LF[i]) {
			float tmp_num = d_num_LF[i];
			float tmp = tmp_num / thread_size;
			int count = ceil(tmp);
			for (int j = 0; j < count; j++) {
				if (tx < d_num_LF[i]) {
					int x = d_LF[len + tx].x;
					int y = d_LF[len + tx].y;
					int z = d_LF[len + tx].z;
					double dot = 0;
					for (int n = 0; n < r; n++) {
						dot += dA[x * r + n] * dB[y * r + n] * dC[z * r + n];
					}
					double error = d_LF[len + tx].rate - dot;

					for (int n = 0; n < r; n++) {
						dA[x * r + n] += lr * (error * dB[y * r + n] * dC[z * r + n] - reg * dA[x * r + n]);
						dB[y * r + n] += lr * (error * dA[x * r + n] * dC[z * r + n] - reg * dB[y * r + n]);
						dC[z * r + n] += lr * (error * dB[y * r + n] * dA[x * r + n] - reg * dC[z * r + n]);
					}
				}
				tx = tx + thread_size;
			}
		}
		else {
			if (tx < d_num_LF[i]) {
				int x = d_LF[len + tx].x;
				int y = d_LF[len + tx].y;
				int z = d_LF[len + tx].z;
				double dot = 0;
				for (int n = 0; n < r; n++) {
					dot += dA[x * r + n] * dB[y * r + n] * dC[z * r + n];
				}
				double error = d_LF[len + tx].rate - dot;

				for (int n = 0; n < r; n++) {
					dA[x * r + n] += lr * (error * dB[y * r + n] * dC[z * r + n] - reg * dA[x * r + n]);
					dB[y * r + n] += lr * (error * dA[x * r + n] * dC[z * r + n] - reg * dB[y * r + n]);
					dC[z * r + n] += lr * (error * dB[y * r + n] * dA[x * r + n] - reg * dC[z * r + n]);
				}
			}
		}
		len += d_num_LF[i];
	}

}
__global__ void tensor_pre_1(LF_node* d_pre, int* d_bsize, t_node* d_tsize, int* d_t_bnum, int* d_t_block_p, double* dA, double* dB, double* dC, int num_parallel, int thread_size, int I, int J, int K, int r, double lr, double reg) {
	int len = 0;
	int len_b = 0;
	int b_id = 0;
	printf("%d \n", d_pre[0].rate);
	for (int i = 0; i < num_parallel; i++) {
		int bx = blockIdx.x;

		if (bx < d_bsize[i]) {
			bx = bx + b_id;
			int t_id = 0;

			for (int j = 0; j < d_tsize[bx].num_b; j++) {
				int tx = threadIdx.x;
				if (tx < d_tsize[bx].num_p) {
					tx += t_id;

					printf("%d %d %d %d %f \n", i, bx, tx, d_t_bnum[bx], d_pre[0].rate);
					if (d_pre[d_t_bnum[bx] + tx].rate > 0) {
						//printf("%d %d %d %d\n", i, tx, len_b, d_pre[d_t_bnum[bx] + tx].rate);
						int x = d_pre[d_t_bnum[bx] + tx].x;
						int y = d_pre[d_t_bnum[bx] + tx].y;
						int z = d_pre[d_t_bnum[bx] + tx].z;
						double dot = 0;
						for (int n = 0; n < r; n++) {
							dot += dA[x * r + n] * dB[y * r + n] * dC[z * r + n];
						}
						double error = d_pre[d_t_bnum[bx] + tx].rate - dot;

						for (int n = 0; n < r; n++) {
							dA[x * r + n] += lr * (error * dB[y * r + n] * dC[z * r + n] - reg * dA[x * r + n]);
							dB[y * r + n] += lr * (error * dA[x * r + n] * dC[z * r + n] - reg * dB[y * r + n]);
							dC[z * r + n] += lr * (error * dB[y * r + n] * dA[x * r + n] - reg * dC[z * r + n]);
						}
					}
				}
				t_id += d_tsize[bx].num_p;
			}
			b_id += d_bsize[i];
		}
	}
}

//preproccess
__global__ void tensor_pre(LF_node* d_pre, int* d_num_parallel_pre, double* dA, double* dB, double* dC, int num_parallel_t, int thread_size, int I, int J, int K, int r, double lr, double reg) {
	int len = 0;
	for (int i = 0; i < num_parallel_t; i++) {
		if (thread_size < d_num_parallel_pre[i]) {
			int tx = threadIdx.x;
			//printf("%d ", d_num_parallel_pre[i]);
			float tmp_num = d_num_parallel_pre[i];
			float tmp = tmp_num / thread_size;
			int count = ceil(tmp);
			for (int j = 0; j < count; j++) {

				if (tx < d_num_parallel_pre[i]) {
					//printf("%d %d %d %d   ",tx, x, y, z);

					int x = d_pre[len + tx].x;
					int y = d_pre[len + tx].y;
					int z = d_pre[len + tx].z;
					double dot = 0;
					for (int n = 0; n < r; n++) {
						dot += dA[x * r + n] * dB[y * r + n] * dC[z * r + n];
					}
					double error = d_pre[len + tx].rate - dot;
					double coe_x = d_pre[len + tx].coe_x;
					double coe_y = d_pre[len + tx].coe_y;
					double coe_z = d_pre[len + tx].coe_z;

					for (int n = 0; n < r; n++) {
						dA[x * r + n] += coe_x * lr * (error * dB[y * r + n] * dC[z * r + n] - reg * dA[x * r + n]);
						dB[y * r + n] += coe_y * lr * (error * dA[x * r + n] * dC[z * r + n] - reg * dB[y * r + n]);
						dC[z * r + n] += coe_z * lr * (error * dB[y * r + n] * dA[x * r + n] - reg * dC[z * r + n]);
					}


				}
				tx = tx + thread_size;
			}
		}
		else {
			int tx = threadIdx.x;
			if (tx < d_num_parallel_pre[i]) {
				//printf("%d ", d_num_parallel_pre[i]);

				int x = d_pre[len + tx].x;
				int y = d_pre[len + tx].y;
				int z = d_pre[len + tx].z;

				//printf("%d %d %d %d %f\n   ",tx, x, y, z, d_pre[len + tx].rate);


				double dot = 0;
				for (int n = 0; n < r; n++) {
					dot += dA[x * r + n] * dB[y * r + n] * dC[z * r + n];
				}
				double error = d_pre[len + tx].rate - dot;
				double coe_x = d_pre[len + tx].coe_x;
				double coe_y = d_pre[len + tx].coe_y;
				double coe_z = d_pre[len + tx].coe_z;

				for (int n = 0; n < r; n++) {
					dA[x * r + n] += coe_x * lr * (error * dB[y * r + n] * dC[z * r + n] - reg * dA[x * r + n]);
					dB[y * r + n] += coe_y * lr * (error * dA[x * r + n] * dC[z * r + n] - reg * dB[y * r + n]);
					dC[z * r + n] += coe_z * lr * (error * dB[y * r + n] * dA[x * r + n] - reg * dC[z * r + n]);
				}

			}
		}
		len += d_num_parallel_pre[i];
	}

}
void clear(queue<double>& q) {
	queue<double> empty;
	swap(empty, q);
}

void sgd_train(double* t_1,
	double* t,
	double* a,
	double* b,
	double* c,
	LF_node* pre,
	int* num_parallel_pre,
	int num_parallel_t,
	int num_block,
	int num_parallel,
	int max_parallel,
	LF_node* LF,
	int* num_LF,
	int nnz,
	double rate,
	int* num_bs,
	b_node* bs,
	Node_conflict* B_conf) {

	ofstream ofs_time_1, ofs_error;
	string address("C:/Users/12625/Desktop/tensor/A_");
	string str;
	stringstream ss;
	ss << rate;
	ss >> str;
	str += "/";
	address += str;
	string add_error, add_time, add_block;
	add_error += address;
	add_error += "error.txt";
	add_time += address;
	add_time += "time.txt";
	add_block += address;
	add_block += "block.txt";

	double* dA, * dB, * dC;
	double* dX;

	int nbytesX = I * J * K * (sizeof(double));
	int nbytesA = I * r * (sizeof(double));
	int nbytesB = J * r * (sizeof(double));
	int nbytesC = K * r * (sizeof(double));

	hipMalloc((void**)&dX, nbytesX);
	hipMalloc((void**)&dA, nbytesA);
	hipMalloc((void**)&dB, nbytesB);
	hipMalloc((void**)&dC, nbytesC);
	if (dA == NULL || dB == NULL || dC == NULL) {
		printf("couldn't allocate GPU memory\n");
	}

	hipMemcpy(dX, t, nbytesX, hipMemcpyHostToDevice);
	hipMemcpy(dA, a, nbytesA, hipMemcpyHostToDevice);
	hipMemcpy(dB, b, nbytesB, hipMemcpyHostToDevice);
	hipMemcpy(dC, c, nbytesC, hipMemcpyHostToDevice);

	clear(num1);

	ofs_error.open(add_error, ios::out | ios::in | ios::trunc);
	ofs_time_1.open(add_time, ios::out | ios::in | ios::app);

	if (flag_lockfree == 1) {
		LF_node* d_LF;
		int* d_num_LF;
		int nbytesLF = nnz * (sizeof(LF_node));
		int nbytesnumLF = num_parallel * (sizeof(int));
		hipMalloc((void**)&d_LF, nbytesLF);
		hipMalloc((void**)&d_num_LF, nbytesnumLF);

		hipMemcpy(d_LF, LF, nbytesLF, hipMemcpyHostToDevice);
		hipMemcpy(d_num_LF, num_LF, nbytesnumLF, hipMemcpyHostToDevice);

		for (int epoch = 0; epoch < epochs; epoch++) {

			double td2 = get_time_2();
			tensor_LF << <1, thread_size >> > (d_LF, d_num_LF, dA, dB, dC, num_parallel, thread_size, I, J, K, r, lr, reg);
			td2 = get_time_2() - td2;
			ofs_time_1 << td2 << endl;

			hipMemcpy(a, dA, nbytesA, hipMemcpyDeviceToHost);
			hipMemcpy(b, dB, nbytesB, hipMemcpyDeviceToHost);
			hipMemcpy(c, dC, nbytesC, hipMemcpyDeviceToHost);

			double Error_1 = RMSE_GPU_S(t, a, b, c);
			double Error_2 = RMSE_GPU_T(t, t_1, a, b, c);
			ofs_error << Error_1 << " " << Error_2 << endl;
			cout << "Epoch:" << epoch << ' ' << Error_1 << " " << Error_2 << '\n';

			num1.push(Error_1);
			if (num1.size() > 2) {
				num1.pop();
			}

			if (num1.size() >= 2) {
				if (num1.front() - num1.back() < 0.0000001) {
					break;
				}
			}
		}

		hipFree(d_LF);
		hipFree(d_num_LF);
	}
	/*
	if (flag_preproccess == 1) {
		
		b_node* d_bs;
		int* d_num_bs;
		Node_conflict* d_B_conf;
		int nbytesbs = nnz * (sizeof(b_node));
		int nbytesnumbs = num_parallel * (sizeof(int));
		int  nbytesconf = num_block * (sizeof(Node_conflict));
		hipMalloc((void**)&d_bs, nbytesbs);
		hipMalloc((void**)&d_num_bs, nbytesnumbs);
		hipMalloc((void**)&d_B_conf, nbytesconf);

		hipMemcpy(d_bs, bs, nbytesbs, hipMemcpyHostToDevice);
		hipMemcpy(d_num_bs, num_bs, nbytesnumbs, hipMemcpyHostToDevice);
		hipMemcpy(d_B_conf, B_conf, nbytesconf, hipMemcpyHostToDevice);

		for (int epoch = 0; epoch < epochs; epoch++) {
			int len_1 = 0;

			double td4 = get_time_2();
			tensor_GPU << <max_num, thread_size >> > (dX, d_bs, d_num_bs, dA, dB, dC, num_parallel, I, J, K, r, lr, reg, d_B_conf);
			td4 = get_time_2() - td4;
			ofs_time_1 << td4 << endl;

			hipMemcpy(a, dA, nbytesA, hipMemcpyDeviceToHost);
			hipMemcpy(b, dB, nbytesB, hipMemcpyDeviceToHost);
			hipMemcpy(c, dC, nbytesC, hipMemcpyDeviceToHost);

			double Error_1 = RMSE_GPU_S(t, a, b, c);
			double Error_2 = RMSE_GPU_T(t, t_1, a, b, c);
			ofs_error << Error_1 << " " << Error_2 << endl;
			cout << "Epoch:" << epoch << ' ' << Error_1 << " " << Error_2 << '\n';

			num1.push(Error_1);
			if (num1.size() > 2) {
				num1.pop();
			}

			if (num1.size() >= 2) {
				if (num1.front() - num1.back() < 0.0000001) {
					break;
				}
			}
		}

		

	}*/

	if (flag_preproccess == 1) {
		LF_node* d_pre;
		int* d_num_parallel_pre;
		int nbytespre = nnz * (sizeof(LF_node));
		int nbytesnumpre = num_parallel_t * (sizeof(int));
		hipMalloc((void**)&d_pre, nbytespre);
		hipMalloc((void**)&d_num_parallel_pre, nbytesnumpre);

		hipMemcpy(d_pre, pre, nbytespre, hipMemcpyHostToDevice);
		hipMemcpy(d_num_parallel_pre, num_parallel_pre, nbytesnumpre, hipMemcpyHostToDevice);

		for (int epoch = 0; epoch < epochs; epoch++) {
			int len_1 = 0;

			double td4 = get_time_2();
			tensor_GPU <<<max_parallel, thread_size >>> (d_pre, d_num_parallel_pre, dA, dB, dC, num_parallel_t, thread_size, I, J, K, r, lr, reg);
			td4 = get_time_2() - td4;
			ofs_time_1 << td4 << endl;

			hipMemcpy(a, dA, nbytesA, hipMemcpyDeviceToHost);
			hipMemcpy(b, dB, nbytesB, hipMemcpyDeviceToHost);
			hipMemcpy(c, dC, nbytesC, hipMemcpyDeviceToHost);

			double Error_1 = RMSE_GPU_S(t, a, b, c);
			double Error_2 = RMSE_GPU_T(t, t_1, a, b, c);
			ofs_error << Error_1 << " " << Error_2 << endl;
			cout << "Epoch:" << epoch << ' ' << Error_1 << " " << Error_2 << '\n';

			num1.push(Error_1);
			if (num1.size() > 2) {
				num1.pop();
			}

			if (num1.size() >= 2) {
				if (num1.front() - num1.back() < 0.0000001) {
					break;
				}
			}
		}

		hipFree(d_pre);

	}
	/*if (flag_preproccess == 1) {
		LF_node* d_pre;
		int* d_bsize;
		t_node* d_tsize;
		int* d_t_bnum;
		int* d_t_block_p;
		int nbytespre = total * (sizeof(LF_node));
		int nbytesnumb = num_parallel * (sizeof(int));
		int nbytesnumt = num_block * (sizeof(t_node));
		int nbytesTbnum = num_block * (sizeof(int));
		int nbytesTbp = num_parallel * (sizeof(int));
		hipMalloc((void**)&d_pre, nbytespre);
		hipMalloc((void**)&d_bsize, nbytesnumb);
		hipMalloc((void**)&d_tsize, nbytesnumt);
		hipMalloc((void**)&d_t_bnum, nbytesTbnum);
		hipMalloc((void**)&d_t_block_p, nbytesTbp);

		hipMemcpy(d_pre, pre, nbytespre, hipMemcpyHostToDevice);
		hipMemcpy(d_bsize, b_size, nbytesnumb, hipMemcpyHostToDevice);
		hipMemcpy(d_tsize, t_size, nbytesnumt, hipMemcpyHostToDevice);
		hipMemcpy(d_t_bnum, t_block_num, nbytesTbnum, hipMemcpyHostToDevice);
		hipMemcpy(d_t_block_p, t_block_p, nbytesTbp, hipMemcpyHostToDevice);

		for (int epoch = 0; epoch < epochs; epoch++) {

			double td4 = get_time_2();
			tensor_pre << <max_parallel, thread_size >> > (d_pre, d_bsize, d_tsize, d_t_bnum, d_t_block_p, dA, dB, dC, num_parallel, thread_size, I, J, K, r, lr, reg);
			int len_1 = 0;
			td4 = get_time_2() - td4;
			ofs_time_1 << td4 << endl;

			hipMemcpy(a, dA, nbytesA, hipMemcpyDeviceToHost);
			hipMemcpy(b, dB, nbytesB, hipMemcpyDeviceToHost);
			hipMemcpy(c, dC, nbytesC, hipMemcpyDeviceToHost);

			double Error_1 = RMSE_GPU_S(t, a, b, c, I, J, K);
			double Error_2 = RMSE_GPU_T(t, t_1, a, b, c, I, J, K);
			ofs_error << Error_1 << " " << Error_2 << endl;
			cout << "Epoch:" << epoch << ' ' << Error_1 << " " << Error_2 << '\n';

			num1.push(Error_1);
			if (num1.size() > 2) {
				num1.pop();
			}

			if (num1.size() >= 2) {
				if (num1.front() - num1.back() < 0.0000001) {
					break;
				}
			}
		}

		hipFree(d_pre);
		hipFree(d_bsize);
		hipFree(d_tsize);
	}*/

	ofs_time_1.close();
	hipFree(dX);
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

}