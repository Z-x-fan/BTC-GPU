#include "hip/hip_runtime.h"
#include <>
#include <hip/hip_runtime.h>
#include<iostream>
#include <vector>
#include<Windows.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include<queue>
#include <bitset>
#include<list>
#include<ctime>
#include <random>
#include <fstream>
#include "tensor_SGD.h"
#include "sgd_Kernel.h"
#include "block.h"
#include <sstream>

using namespace std;

int I = 144;
int J = 288;
int K = 168;

int epochs = 1000;
int r = 4;
double lr = 0.0002;
double reg = 0.02;
int block_s = 4096;     //block�����������
//float sample_rate = (float)0.5;
int thread_size = 256;
int flag_lockfree = 0;
int flag_preproccess = 1;
int max_num = 8;
queue<double> num1;

//������ɾ���
void getRand(double* Mat, int I, int r)
{
	for (int i = 0; i < I; i++)
	{
		for (int j = 0; j < r; j++)
		{
			random_device rd;
			mt19937 gen(rd());
			uniform_real_distribution<> distribution(0.1, 0.9);
			double random = distribution(gen);
			Mat[i * r + j] = random;

		}
	}

}
//ȡ��
void sample(double* t, double* s, int* t_s, float num_p, int I, int J, int K) {

	int num_z = floor(num_p * K);
	for (int i = 0; i < I; i++) {
		for (int j = 0; j < J; j++) {
			int num = 0;
			int* tmp = new int[K]();
			while (num < num_z)
			{
				random_device rd;
				mt19937 gen(rd());
				uniform_real_distribution<> distribution(0, K * 1.1);
				int k = distribution(gen);

				if (k < K && tmp[k] == 0) {
					tmp[k] = 1;
					t_s[i * J * K + j * K + k] = 1;
					num++;
				}
			}
		}
	}

	for (int i = 0; i < I; i++) {
		for (int j = 0; j < J; j++) {
			for (int m = 0; m < K; m++) {
				t[i * J * K + j * K + m] = t[i * J * K + j * K + m] * t_s[i * J * K + j * K + m];

			}
		}
	}

}
vector<vector<vector<double>>> sample_v(vector<vector<vector<double>>> S, double* t, float num_p, int I, int J, int K) {
	int num_z = floor(num_p * K);
	for (int i = 0; i < I; i++) {
		for (int j = 0; j < J; j++) {
			int num = 0;
			while (num < num_z)
			{
				random_device rd;
				mt19937 gen(rd());
				uniform_real_distribution<> distribution(0, K * 1.1);
				int k = distribution(gen);

				if (k < K && S[i][j][k] == 0) {
					S[i][j][k] = t[i * J * K + j * K + k];
					num++;
				}
			}
		}
	}

	return S;
}
int sample_vary(double* s, double* t, int* t_s, float num_p, int I, int J, int K) {
	int num_z = floor(num_p * K);
	int num = 0;
	int nnz = 0;
	while (num < num_z) {
		random_device rd;
		mt19937 gen(rd());
		uniform_real_distribution<> distribution_x(0, I - 1);
		int x = distribution_x(gen);
		uniform_real_distribution<> distribution_y(0, J - 1);
		int y = distribution_y(gen);
		uniform_real_distribution<> distribution_z(0, K - 1);
		int z = distribution_z(gen);

		if (t_s[x * J * K + y * K + z] == 0) {
			t_s[x * J * K + y * K + z] = 1;
			num++;
		}
	}

	for (int i = 0; i < I * J * K; i++) {
		s[i] = t[i] * t_s[i];
	}
	for (int i = 0; i < I * J * K; i++) {
		if (s[i] > 0)
			nnz++;
	}
	return nnz;
}

//����SתΪһά���飬��¼�������
int Create_t(double* t, int I, int J, int K) {
	int nnz = 0;
	for (int i = 0; i < I; i++) {
		for (int j = 0; j < J; j++) {
			for (int m = 0; m < K; m++) {
				if (t[i * J * K + j * K + m] > 0) {
					nnz++;
				}
				else {
					printf("%d %d %d %f \n", i, j, m, t[i * J * K + j * K + m]);
				}
			}
		}
	}
	return nnz;
}

//�������
int Create_tmp( double* t, int I, int J, int K) {
	int nnz = 0;
	for (int i = 0; i < I; i++) {
		for (int j = 0; j < J; j++) {
			for (int m = 0; m < K; m++) {
				if (t[i * J * K + j * K + m] > 0)
					nnz++;
			}
		}
	}
	return nnz;
}

//���ݹ�һ��
void Normalization(double* t_2, double* t_1, double& max, double& min) {
	max = min = t_2[0];
	for (int i = 0; i < I * J * K; i++) {
		max = max > t_2[i] ? max : t_2[i];
		min = min < t_2[i] ? min : t_2[i];
	}
	for (int i = 0; i < I * J * K; i++) {
		t_1[i] = 0.01 + 0.99 * (t_2[i] - min) / (max - min);
	}
}

void tensor_SGD(double* t_2) {
	
	double* t_1 = new double[I * J * K]();

	t_1 = t_2;

	double rate = 0.5;
	//	while (rate <= 1) {
	printf("before: %f %f %f %f\n", t_1[0], t_1[1], t_1[(I - 1) * J * K + (J - 1) * K + (K - 1) - 1], t_1[(I - 1) * J * K + (J - 1) * K + (K - 1)]);
	

	//���ļ���������������
	double* a = new double[I * r];
	double* b = new double[J * r];
	double* c = new double[K * r];
	getRand(a, I, r);
	getRand(b, J, r);
	getRand(c, K, r);

	cout << rate << endl;
	double* t = new double[I * J * K];
	string address_data("C:/Users/12625/Desktop/tensor/AbileneTM_Tensor_");
	string str;
	stringstream ss;
	ss << rate;
	ss >> str;
	str += ".txt";
	address_data += str;

	ifstream infile(address_data);
	int sum = 0;
	while (infile) {
		infile >> t[sum];
		sum++;
	}

	int nnz = 0;

	//S = sample(t_1, rate, I, J, K);
	//S = sample_vary(S, t_1, 0.1, I, J, K);
	//nnz = Create_t(S, t, I, J, K);
	nnz = Create_tmp(t, I, J, K);
	//t = t_s;
	//nnz=Create_t(t, I, J, K);
	 
	/*if (rate < 1) {
		//S = sample(t_1, rate, I, J, K);
		//S = sample_vary(S, t_1, 0.1, I, J, K);
		//nnz = Create_t(S, t, I, J, K);
		nnz = Create_tmp(t_s, t, t_1, I, J, K);
		//t = t_s;
		//nnz=Create_t(t, I, J, K);
	}
	else {
		t = t_1;
		t_s = t_1;
		//nnz = Create_tmp(t, I, J, K);
		nnz = Create_t(t, I, J, K);
	}*/

	/*		ofstream ofs_s;
			ofs_s.open("C:\\Users\\12625\\Desktop\\tensor\\test_s.txt", ios::out | ios::in | ios::trunc);

			for (int i = 0; i < I; i++) {
				for (int j = 0; j < J; j++) {
					for (int m = 0; m < K; m++) {

						ofs_s << S[i][j][m] << " ";
					}
				}
			}
			ofs_s.close();*/

	LF_node* pre = NULL;
	int* num_parallel_pre = NULL;
	int num_block = 0;//block������
	int num_parallel = 0;//��������
	int max_parallel = 0;//�����block��
	LF_node* LF = NULL;
	int* num_LF = NULL;


	block_problem(t_1, t, a, b, c, pre, num_parallel_pre, num_block, num_parallel, max_parallel, LF, num_LF, nnz, rate);

	double* t_3 = new double[I * J * K];
	for (int i = 0; i < I; i++) {
		for (int j = 0; j < J; j++) {
			for (int m = 0; m < K; m++) {
				double sum_t_1 = 0;
				for (int n = 0; n < r; n++) {
					sum_t_1 += a[i * r + n] * b[j * r + n] * c[m * r + n];
				}
				t_3[i * J * K + j * K + m] = sum_t_1;
			}
		}
	}

	printf("after: %f %f %f %f\n", t_3[0], t_3[1], t_3[(I - 1) * J * K + (J - 1) * K + (K - 1) - 1], t_3[(I - 1) * J * K + (J - 1) * K + (K - 1)]);
	delete[]a;
	delete[]b;
	delete[]c;

	//		delete []t;

	delete LF;
	delete num_LF;
	delete num_parallel_pre;
	delete[]t_3;
	rate += 0.1;
	//}

	delete[]t_1;
	/*
		double* a = new double[I * r];
		double* b = new double[J * r];
		double* c = new double[K * r];
		getRand(a, I, r);
		getRand(b, J, r);
		getRand(c, K, r);

		vector<vector<vector<double>>> S(I, vector<vector<double>>(J, vector<double>(K, 0)));
		double* t = new double[I * J * K];
		int nnz = 0;
		if (sample_rate < 1) {
			S = sample(t_1, 0.5, I, J, K);
			nnz = Create_t(S, t, I, J, K);
		}
		else {
			t = t_1;
			nnz = I * J * K;
		}

		b_node* bs = NULL;
		int* num_bs = NULL;
		int num_block;
		int num_parallel;
		int max_parallel;
		LF_node* LF = NULL;
		int* num_LF = NULL;
		LF_node* pre_t = NULL;
		int num_parallel_t;
		int* num_parallel_pre = NULL;

		block_problem(t_1, t, a, b, c, bs, num_bs, num_block, num_parallel, max_parallel, LF, num_LF, pre_t, num_parallel_t, num_parallel_pre, nnz, rate);

		for (int i = 0; i < I; i++) {
			for (int j = 0; j < J; j++) {
				for (int m = 0; m < K; m++) {
					for (int n = 0; n < r; n++) {
						t_1[i * J * K + j * K + m] = a[i * r + n] * b[j * r + n] * c[m * r + n];
					}

				}
			}
		}*/

		/*
			double* t_2 = new double[I * J * K];
			ofstream ofs;
			ofs.open("C:/Users/12625/Desktop/tensor/tensor_after.txt", ios::out | ios::in);
			for (int i = 0; i < I; i++) {
				for (int j = 0; j < J; j++) {
					for (int m = 0; m < K; m++) {
						double sum_t_1 = 0;
						for (int n = 0; n < r; n++) {
							sum_t_1 += a[i * r + n] * b[j * r + n] * c[m * r + n];
						}
						t_2[i * J * K + j * K + m] = sum_t_1;
						ofs << t_2[i * J * K + j * K + m] << " ";

					}
				}
			}
			ofs.close();*/

}



