#include "hip/hip_runtime.h"
#include <>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <set>
#include <Windows.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <queue>
#include <bitset>
#include <list>
#include <ctime>
#include <random>
#include <fstream>
#include <sstream>
#include <cstdlib>
#include "tensor_SGD.h"
#include "block.h"
#include "sgd_Kernel.h"


using namespace std;

//��ȡʱ��
double get_time(void)
{
	LARGE_INTEGER timer;
	static LARGE_INTEGER fre;
	static int init = 0;
	double t;

	if (init != 1) {
		QueryPerformanceFrequency(&fre);
		init = 1;
	}

	QueryPerformanceCounter(&timer);

	t = timer.QuadPart * 1. / (double)fre.QuadPart;

	return t;
}

//ȡ��������
void initial_t(T_node* t_block, double* t, int I, int J, int K, int nnz) {
	int num = 0;

	for (int m = 0; m < K; m++) {
		for (int j = 0; j < J; j++) {
			for (int i = 0; i < I; i++) {
				if (t[i * J * K + j * K + m] > 0) {
					t_block[num].x = i;
					t_block[num].y = j;
					t_block[num].z = m;
					num++;
				}
			}
		}
	}
	random_device rd;
	mt19937 gen(rd());
	uniform_real_distribution<> distribution(0, nnz * 0.6);
	int random = distribution(gen);

	int size = random;
	for (int i = size; i < nnz; i++) {
		uniform_real_distribution<> distribution(0, nnz);
		int random = distribution(gen);
		if (random < size) {
			T_node tmp_t;
			tmp_t.x = t_block[i].x;
			tmp_t.y = t_block[i].y;
			tmp_t.z = t_block[i].z;
			t_block[i].x = t_block[random].x;
			t_block[i].y = t_block[random].y;
			t_block[i].z = t_block[random].z;
			t_block[random].x = tmp_t.x;
			t_block[random].y = tmp_t.y;
			t_block[random].z = tmp_t.z;
		}
	}
}

//���껯Ϊ������
vector<bool> hx(int end, int start, int level, int num) {
	vector<bool> binary;

	float n_1 = (end - start + 1) / pow(2, level - 1);
	int n_2 = floor(num / n_1);

	bitset<100> b1(n_2);

	for (int i = level - 1; i >= 0; i--) {
		binary.push_back(b1[i]);
	}
	return binary;
	
	/*
	bitset<100> b1(n_2);
	int* bit = new int[level];
	int num = 0;
	for (int i = level - 1; i >= 0; i--) {
		bit[num] = b1[i];
		num++;
	}
	return bit;
	*/

}

//�����������Ƿ�ƥ��
int compare(vector<bool> id, vector<bool> x, int level) {
	for (int block_id = 1; block_id < level; block_id++) {
		if (x[block_id] ^ id[block_id]) {   //���  ��ͬΪ1����ͬΪ0
			return 1;   //����1Ϊ����ͻ
		}
	}
	return 0;// id��ͬ
/*
	for (int block_id = 0; block_id < level; block_id++) {
		if (x[block_id] != id[block_id]) {
			flag = 1;
			break;
		}
	}

	if (flag == 0)
		return 0;
	else if (flag == 1)
		return 1;
	else
		return false;*/
}

//�ҵ�������ƥ���BlockID
BTnode* search_block(vector<bool> x, vector<bool> y, vector<bool> z, BTnode* BT) {
	while (BT->lChild != NULL) {
		int tmp_h = BT->h % 3;
		if (tmp_h == 1) {		
			if (compare(BT->lChild->x, x, BT->lChild->x.size()) == 0)
				BT = BT->lChild;
			else
				BT = BT->rChild;
		}
		else if (tmp_h == 2) {			
			if (compare(BT->lChild->y, y, BT->lChild->y.size()) == 0)
				BT = BT->lChild;
			else
				BT = BT->rChild;
		}
		else {
			if (compare(BT->lChild->z, z, BT->lChild->z.size()) == 0)
				BT = BT->lChild;
			else
				BT = BT->rChild;
		}
	}
/*
	vector<bool> tmp_x = x;
	vector<bool> tmp_y = y;
	vector<bool> tmp_z = z;
	tmp_x.erase(tmp_x.begin());
	tmp_y.erase(tmp_y.begin());
	tmp_z.erase(tmp_z.begin());
	while (BT->lChild != NULL) {
		int tmp_h = BT->h % 3;
		if (tmp_h == 1) {
			int s_x = tmp_x.front();
			tmp_x.erase(tmp_x.begin());
			if (s_x == 0)
				BT = BT->lChild;
			else
 				BT = BT->rChild;
		}
		else if (tmp_h == 2) {
			int s_y = tmp_y.front();
			tmp_y.erase(tmp_y.begin());
			if (s_y == 0)
				BT = BT->lChild;
			else
				BT = BT->rChild;
		}
		else {
			int s_z = tmp_z.front();
			tmp_z.erase(tmp_z.begin());
			if (s_z == 0)
				BT = BT->lChild;
			else
				BT = BT->rChild;
		}
	}
*/	
	return BT;
}
/*
int search_block(BS_node* Block, int* x, int* y, int* z, int block_id_num) {
	
	int search_id = -1;
	int flag = 1;
	for (int i = 0; i < block_id_num; i++) {
		int flag = compare(Block[i].x_id, x, Block[i].level_x);
		if (flag == 0) {
			flag = compare(Block[i].y_id, y, Block[i].level_y);
		}
		if (flag == 0) {
			flag = compare(Block[i].z_id, z, Block[i].level_z);
		}
		if (flag == 0) {
			search_id = i;
			break;
		}
	}
	return search_id;
}*/

void Create_Btree(BTnode* BT_block,int old_id, int new_id) {
	BT_block->block = -1;
	int tmp_h = BT_block->h % 3;

	if (tmp_h == 1) {
		BTnode* s_l = new BTnode();
		BTnode* s_r = new BTnode();
		s_l->h = BT_block->h + 1;
		s_r->h = BT_block->h + 1;
		s_l->x = BT_block->x;
		s_l->y = BT_block->y;
		s_l->z = BT_block->z;
		s_l->x.push_back(0);
		s_r->x = BT_block->x;
		s_r->y = BT_block->y;
		s_r->z = BT_block->z;
		s_r->x.push_back(1);
		s_l->block = old_id;
		s_r->block = new_id;
		s_l->lChild = s_l->rChild = s_r->lChild = s_r->rChild = NULL;
		BT_block->lChild = s_l;
		BT_block->rChild = s_r;
	}
	else if (tmp_h == 2) {
		BTnode* s_l = new BTnode();
		BTnode* s_r = new BTnode();
		s_l->h = BT_block->h + 1;
		s_r->h = BT_block->h + 1;
		s_l->x = BT_block->x;
		s_l->y = BT_block->y;
		s_l->z = BT_block->z;
		s_l->y.push_back(0);
		s_r->x = BT_block->x;
		s_r->y = BT_block->y;
		s_r->z = BT_block->z;
		s_r->y.push_back(1);
		s_l->block = old_id;
		s_r->block = new_id;
		s_l->lChild = s_l->rChild = s_r->lChild = s_r->rChild = NULL;
		BT_block->lChild = s_l;
		BT_block->rChild = s_r;
	}
	else if (tmp_h == 0) {
		BTnode* s_l = new BTnode();
		BTnode* s_r = new BTnode();
		s_l->h = BT_block->h + 1;
		s_r->h = BT_block->h + 1;
		s_l->x = BT_block->x;
		s_l->y = BT_block->y;
		s_l->z = BT_block->z;
		s_l->z.push_back(0);
		s_r->x = BT_block->x;
		s_r->y = BT_block->y;
		s_r->z = BT_block->z;
		s_r->z.push_back(1);
		s_l->block = old_id;
		s_r->block = new_id;
		s_l->lChild = s_l->rChild = s_r->lChild = s_r->rChild = NULL;
		BT_block->lChild = s_l;
		BT_block->rChild = s_r;
	}

}
/*
int* ID_change(int* id, int number, int level) {
	int* s = new int[level];
	for (int i = 0; i < level - 1; i++) {
		s[i] = id[i];
	}
	s[level - 1] = number;
	return s;
}

//�ж��Ƿ�Ҫ�����ֿ�
int continue_partition(int* t, int* id, int* x, int level, int end, int start, int g) {
	int flag = 1;
	int num = 0;
	flag = compare(id, x, level);
	if (flag == 0)
		num++;
	for (int i = 0; i < block_s; i++) {
		int* Bx = hx(end, start, g, t[i]);
		flag = compare(id, Bx, level);
		if (flag == 0)
			num++;
	}
	if (num == block_s + 1 || num == 0)
		return 0;
	else
		return 1;
}*/

//���ݿ黮��
int tensor_block(BS_node* Block, T_node* t, int nnz, int I, int J, int K, BTnode* BT, BTnode* BT_head) {
	int Block_num = 1;//Block����
	BT->h = 1;
	BT->block = 0;
	BT->x.push_back(0);
	BT->y.push_back(0);
	BT->z.push_back(0);
	BT->lChild = BT->rChild = NULL;	
	int g_x, g_y, g_z;
	g_x = 1;
	g_y = 1;
	g_z = 1;
	int start = 0;
	int end_x = I;
	int end_y = J;
	int end_z = K;
	for (int i = 0; i < nnz; i++) {		
		vector<bool> x = hx(end_x, start, g_x, t[i].x);
		vector<bool> y = hx(end_y, start, g_y, t[i].y);
		vector<bool> z = hx(end_z, start, g_z, t[i].z);

		BT = BT_head;
		BT = search_block(x, y, z, BT);
		int id = BT->block;

		if (Block[id].block_num < block_s) {
			Block[id].t[Block[id].block_num].x = t[i].x;
			Block[id].t[Block[id].block_num].y = t[i].y;
			Block[id].t[Block[id].block_num].z = t[i].z;
			Block[id].block_num++;
		}
		else {
			int flag = 0;
			while (flag == 0) {

				if (Block[id].level_x == Block[id].level_y && Block[id].level_y == Block[id].level_z) {
					Block[id].level_x++;
					int max = 0;
					max = Block[id].level_x > g_x ? Block[id].level_x : g_x;
					g_x = max;

					Create_Btree(BT, id, Block_num);
					//Block[block_id_num].x_id = ID_change(Block[id].x_id, 1, Block[id].level_x);
					//Block[id].x_id = ID_change(Block[id].x_id, 0, Block[id].level_x);
					Block[Block_num].block_num = 0;
					Block[Block_num].level_x = Block[id].level_x;
					Block[Block_num].level_y = Block[id].level_y;
					Block[Block_num].level_z = Block[id].level_z;
					Block[Block_num].x_id = Block[id].x_id;
					Block[Block_num].y_id = Block[id].y_id;
					Block[Block_num].z_id = Block[id].z_id;
					Block[Block_num].id = Block_num;
					Block[Block_num].t = new T_node[block_s];
					Block[Block_num].x_id.push_back(1);
					Block[id].x_id.push_back(0);

					//�������·���
					int num_1 = 0;
					int num_2 = 0;
					int* tx = new int[block_s];
					T_node* t1 = new T_node[block_s];
					T_node* t2 = new T_node[block_s];
					for (int j = 0; j < Block[id].block_num; j++) {
						vector<bool> xt = hx(end_x, start, g_x, Block[id].t[j].x);
						//int flag_1 = compare(Block[id].x_id, xt, Block[id].level_x);
						if (xt.back() == 0) {
							t1[num_1].x = Block[id].t[j].x;
							t1[num_1].y = Block[id].t[j].y;
							t1[num_1].z = Block[id].t[j].z;
							num_1++;
						}
						else {
							t2[num_2].x = Block[id].t[j].x;
							t2[num_2].y = Block[id].t[j].y;
							t2[num_2].z = Block[id].t[j].z;
							num_2++;
						}					
					}
					vector<bool> xt = hx(end_x, start, g_x, t[i].x);
					if (num_1 < block_s && num_2 < block_s) {
						flag = 1;
						//int flag_1 = compare(Block[id].x_id, x, Block[id].level_x);
						
						if (xt.back() == 0) {
							t1[num_1].x = t[i].x;
							t1[num_1].y = t[i].y;
							t1[num_1].z = t[i].z;
							num_1++;
						}
						else {
							t2[num_2].x = t[i].x;
							t2[num_2].y = t[i].y;
							t2[num_2].z = t[i].z;
							num_2++;
						}
					}
					else {
						flag = 0;
						if (xt.back() == 0) {
							id = Block[id].id;
							BT = BT->lChild;
						}							
						else {
							id = Block[Block_num].id;
							BT = BT->rChild;
						}							
					}
					

					Block[id].t = t1;
					Block[id].block_num = num_1;
					Block[Block_num].t = t2;
					Block[Block_num].block_num = num_2;
																
					/*flag = continue_partition(tx, Block[id].x_id, x, Block[id].level_x, end_x, start, g_x);

					if (flag == 0) {
						id = search_block(Block, x, y, z, block_id_num);
					}
					else {
						int num_1 = 0;
						int num_2 = 0;
						T_node* t1 = new T_node[block_s];
						T_node* t2 = new T_node[block_s];
						int flag_1 = compare(Block[id].x_id, x, Block[id].level_x);
						if (flag_1 == 0) {
							t1[num_1].x = t[i].x;
							t1[num_1].y = t[i].y;
							t1[num_1].z = t[i].z;
							num_1++;
						}
						else {
							t2[num_2].x = t[i].x;
							t2[num_2].y = t[i].y;
							t2[num_2].z = t[i].z;
							num_2++;
						}
						for (int j = 0; j < Block[id].block_num; j++) {
							int* xt = hx(end_x, start, g_x, Block[id].t[j].x);
							int flag_1 = compare(Block[id].x_id, xt, Block[id].level_x);
							if (flag_1 == 0) {
								t1[num_1].x = Block[id].t[j].x;
								t1[num_1].y = Block[id].t[j].y;
								t1[num_1].z = Block[id].t[j].z;
								num_1++;
							}
							else {
								t2[num_2].x = Block[id].t[j].x;
								t2[num_2].y = Block[id].t[j].y;
								t2[num_2].z = Block[id].t[j].z;
								num_2++;
							}

							delete xt;
						}

						Block[id].t = t1;
						Block[id].block_num = num_1;
						Block[block_id_num].t = t2;
						Block[block_id_num].block_num = num_2;
					}*/
					Block_num++;
				}
				else if (Block[id].level_y < Block[id].level_x) {
					Block[id].level_y++;
					int max = 0;
					max = Block[id].level_y > g_y ? Block[id].level_y : g_y;
					g_y = max;


					Create_Btree(BT, id, Block_num);
					//Block[block_id_num].y_id = ID_change(Block[id].y_id, 1, Block[id].level_y);
					//Block[id].y_id = ID_change(Block[id].y_id, 0, Block[id].level_y);
					Block[Block_num].block_num = 0;
					Block[Block_num].level_x = Block[id].level_x;
					Block[Block_num].level_y = Block[id].level_y;
					Block[Block_num].level_z = Block[id].level_z;
					Block[Block_num].x_id = Block[id].x_id;
					Block[Block_num].y_id = Block[id].y_id;
					Block[Block_num].z_id = Block[id].z_id;
					Block[Block_num].id = Block_num;
					Block[Block_num].t = new T_node[block_s];
					Block[Block_num].y_id.push_back(1);
					Block[id].y_id.push_back(0);


					int* ty = new int[block_s];
					int num_1 = 0;
					int num_2 = 0;
					T_node* t1 = new T_node[block_s];
					T_node* t2 = new T_node[block_s];
					for (int j = 0; j < Block[id].block_num; j++) {
						vector<bool> yt = hx(end_y, start, g_y, Block[id].t[j].y);
						//int flag_1 = compare(Block[id].y_id, yt, Block[id].level_y);
						if (yt.back() == 0) {
							t1[num_1].x = Block[id].t[j].x;
							t1[num_1].y = Block[id].t[j].y;
							t1[num_1].z = Block[id].t[j].z;
							num_1++;
						}
						else {
							t2[num_2].x = Block[id].t[j].x;
							t2[num_2].y = Block[id].t[j].y;
							t2[num_2].z = Block[id].t[j].z;
							num_2++;
						}						                  
					}
					vector<bool> yt = hx(end_y, start, g_y, t[i].y);
					if (num_1 < block_s && num_2 < block_s) {
						flag = 1;
						//int flag_1 = compare(Block[id].x_id, x, Block[id].level_x);
						
						if (yt.back() == 0) {
							t1[num_1].x = t[i].x;
							t1[num_1].y = t[i].y;
							t1[num_1].z = t[i].z;
							num_1++;
						}
						else {
							t2[num_2].x = t[i].x;
							t2[num_2].y = t[i].y;
							t2[num_2].z = t[i].z;
							num_2++;
						}
					}
					else {
						flag = 0;
						if (yt.back() == 0) {
							id = Block[id].id;
							BT = BT->lChild;
						}
						else {
							id = Block[Block_num].id;
							BT = BT->rChild;
						}
					}
					Block[id].t = t1;
					Block[id].block_num = num_1;
					Block[Block_num].t = t2;
					Block[Block_num].block_num = num_2;
					/*y = hx(end_y, start, g_y, t[i].y);
					for (int j = 0; j < Block[id].block_num; j++) {
						ty[j] = Block[id].t[j].y;
					}
					flag = continue_partition(ty, Block[id].y_id, y, Block[id].level_y, end_y, start, g_y);

					if (flag == 0) {
						id = search_block(Block, x, y, z, block_id_num);
					}
					else {
						int num_1 = 0;
						int num_2 = 0;
						T_node* t1 = new T_node[block_s];
						T_node* t2 = new T_node[block_s];
						int flag_1 = compare(Block[id].y_id, y, Block[id].level_y);
						if (flag_1 == 0) {
							t1[num_1].x = t[i].x;
							t1[num_1].y = t[i].y;
							t1[num_1].z = t[i].z;
							num_1++;
						}
						else {
							t2[num_2].x = t[i].x;
							t2[num_2].y = t[i].y;
							t2[num_2].z = t[i].z;
							num_2++;
						}
						for (int j = 0; j < Block[id].block_num; j++) {
							int* yt = hx(end_y, start, g_y, Block[id].t[j].y);
							int flag_1 = compare(Block[id].y_id, yt, Block[id].level_y);
							if (flag_1 == 0) {
								t1[num_1].x = Block[id].t[j].x;
								t1[num_1].y = Block[id].t[j].y;
								t1[num_1].z = Block[id].t[j].z;
								num_1++;
							}
							else {
								t2[num_2].x = Block[id].t[j].x;
								t2[num_2].y = Block[id].t[j].y;
								t2[num_2].z = Block[id].t[j].z;
								num_2++;
							}

							delete yt;
						}

						Block[id].t = t1;
						Block[id].block_num = num_1;
						Block[block_id_num].t = t2;
						Block[block_id_num].block_num = num_2;
					}*/
					Block_num++;
				}
				else if (Block[id].level_z < Block[id].level_y) {

					Block[id].level_z++;
					int max = 0;
					max = Block[id].level_z > g_z ? Block[id].level_z : g_z;
					g_z = max;

					Create_Btree(BT, id, Block_num);
					//Block[block_id_num].z_id = ID_change(Block[id].z_id, 1, Block[id].level_z);
					//Block[id].z_id = ID_change(Block[id].z_id, 0, Block[id].level_z);
					Block[Block_num].block_num = 0;
					Block[Block_num].level_x = Block[id].level_x;
					Block[Block_num].level_y = Block[id].level_y;
					Block[Block_num].level_z = Block[id].level_z;
					Block[Block_num].x_id = Block[id].x_id;
					Block[Block_num].y_id = Block[id].y_id;
					Block[Block_num].z_id = Block[id].z_id;
					Block[Block_num].id = Block_num;
					Block[Block_num].t = new T_node[block_s];
					Block[Block_num].z_id.push_back(1);
					Block[id].z_id.push_back(0);


					int* tz = new int[block_s];
					int num_1 = 0;
					int num_2 = 0;
					T_node* t1 = new T_node[block_s];
					T_node* t2 = new T_node[block_s];					
					for (int j = 0; j < Block[id].block_num; j++) {
						vector<bool> zt = hx(end_z, start, g_z, Block[id].t[j].z);
						//int flag_1 = compare(Block[id].z_id, zt, Block[id].level_z);
						if (zt.back() == 0) {
							t1[num_1].x = Block[id].t[j].x;
							t1[num_1].y = Block[id].t[j].y;
							t1[num_1].z = Block[id].t[j].z;
							num_1++;
						}
						else {
							t2[num_2].x = Block[id].t[j].x;
							t2[num_2].y = Block[id].t[j].y;
							t2[num_2].z = Block[id].t[j].z;
							num_2++;
						}
					}
					vector<bool> zt = hx(end_z, start, g_z, t[i].z);
					if (num_1 < block_s && num_2 < block_s) {
						flag = 1;
						//int flag_1 = compare(Block[id].x_id, x, Block[id].level_x);
						
						if (zt.back() == 0) {
							t1[num_1].x = t[i].x;
							t1[num_1].y = t[i].y;
							t1[num_1].z = t[i].z;
							num_1++;
						}
						else {
							t2[num_2].x = t[i].x;
							t2[num_2].y = t[i].y;
							t2[num_2].z = t[i].z;
							num_2++;
						}
					}
					else {
						flag = 0;
						if (zt.back() == 0) {
							id = Block[id].id;
							BT = BT->lChild;
						}
						else {
							id = Block[Block_num].id;
							BT = BT->rChild;
						}
					}
					Block[id].t = t1;
					Block[id].block_num = num_1;
					Block[Block_num].t = t2;
					Block[Block_num].block_num = num_2;
					Block_num++;
				}
			}
		}

		vector<bool>().swap(x);
		vector<bool>().swap(y);
		vector<bool>().swap(z);
		//delete x;		delete y;		delete z;

	}

	return Block_num;
}

void Free_list(Parallel* head) {

	Parallel* freeNode;
	while (NULL != head) {
		freeNode = head;
		head = head->next;
		delete freeNode;
	}
}

void destroyTree(BTnode* root) {
	if (root == NULL) {
		return;
	}
	destroyTree(root->lChild);
	destroyTree(root->rChild);
	free(root);
}

int compare_parallel(BS_node a, BS_node b) {
	int flag = 1;
	int level = 1;
	level = a.level_x < b.level_x ? a.level_x : b.level_x;
	if (level == 1)
		flag = 1;
	else {
		int block_x;
		for (block_x = 1; block_x < level; block_x++) {
			if (a.x_id[block_x] != b.x_id[block_x]) {
				flag = 0;
				break;
			}
		}
		if (block_x == level)
			return 1;
	}
	if (flag == 0) {
		level = a.level_y < b.level_y ? a.level_y : b.level_y;
		if (level == 1)
			flag = 1;
		else {
			int block_y;
			for (block_y = 1; block_y < level; block_y++) {
				if (a.y_id[block_y] != b.y_id[block_y]) {
					flag = 0;
					break;
				}
			}
			if (block_y == level)
				return 1;
		}
	}
	if (flag == 0) {
		level = a.level_z < b.level_z ? a.level_z : b.level_z;
		if (level == 1)
			flag = 1;
		else {
			int block_z;
			for (block_z = 1; block_z < level; block_z++) {
				if (a.z_id[block_z] != b.z_id[block_z]) {
					flag = 0;
					break;
				}
			}
			if (block_z == level)
				return 1;
		}
	}

	if (flag == 0)//���Բ���
		return 0;
	else if (flag == 1)
		return 1;
	else
		return false;
}

int compare_parallel_2(BS_node a, BS_node b, int num) {
	int flag = 1;
	int level = 1;
	level = a.level_x < b.level_x ? a.level_x : b.level_x;
	if (level == 1)
		flag = 1;
	else {
		int block_x;
		for (block_x = 1; block_x < level; block_x++) {
			if (a.x_id[block_x] != b.x_id[block_x]) {
				flag = 0;
				break;
			}
		}
		if (block_x == level) {
			num++;
			flag == 1;
		}

	}
	if (flag == 0) {
		level = a.level_y < b.level_y ? a.level_y : b.level_y;
		if (level == 1)
			flag = 1;
		else {
			int block_y;
			for (block_y = 1; block_y < level; block_y++) {
				if (a.y_id[block_y] != b.y_id[block_y]) {
					flag = 0;
					break;
				}
			}
			if (block_y == level) {
				num++;
				flag = 1;
			}
		}
	}
	if (flag == 0) {
		level = a.level_z < b.level_z ? a.level_z : b.level_z;
		if (level == 1)
			flag = 1;
		else {
			int block_z;
			for (block_z = 1; block_z < level; block_z++) {
				if (a.z_id[block_z] != b.z_id[block_z]) {
					flag = 0;
					break;
				}
			}
			if (block_z == level) {
				num++;
				flag = 1;
			}
		}
	}

	if (flag == 0)//���Բ���
		return 0;
	else if (flag == 1)
		return 1;
	else
		return false;
}

//Ѱ�Ҳ���Block
Parallel* search_parallel_block_2(BS_node* Block, int block_num, int& num_parallel, int& max_parallel, int& block_pro) {
	num_parallel = 1;
	Parallel* P, * head, * tmp;
	P = new Parallel();
	head = new Parallel();
	tmp = new Parallel();
	P->next = NULL;
	head = P;

	tmp->L.push_back(0);
	tmp->next = P->next;
	P->next = tmp;
	for (int i = 1; i < block_num; i++) {
		int flag = 0;
		P = head;
		while (P->next != NULL) {
			P = P->next;
			int num = P->L.size();
			max_parallel = max_parallel > num ? max_parallel : num;

			list<int>::iterator it = P->L.begin();
			for (it; it != P->L.end(); it++) {
				flag = compare_parallel(Block[i], Block[*it]);
				if (flag == 1)
					break;
			}
			if (flag == 0) {
				P->L.push_back(i);
				break;
			}
		}
		if (flag == 1) {
			num_parallel++;
			Parallel* s;
			s = new Parallel();
			s->L.push_back(i);
			s->next = NULL;
			P->next = s;
			P = s;
		}
	}
	/**/
	P = head;
	list<int> L;
	while (P->next != NULL) {
		P = P->next;
		int num = P->L.size();
		if (num == 1) {
			int number = P->L.front();
			L.push_back(number); 
			P->L.pop_front();
		}
	}
	P = head;
	
	while (P->next != NULL) {
		P = P->next;
		int num = 0;
		if (P->L.size() < max_parallel) {
			list<int>::iterator it = P->L.begin();
			for (it; it != P->L.end(); it++) {
				if (num > 1)
					break;
				else {
					list<int>::iterator it_1 = L.begin();
					for (it_1; it_1 != P->L.end(); it_1++) {
//						compare_parallel_2()
					}
				}
				
			}
			
			for (int i = 0; i < block_num; i++) {
				if (P->L.size() == max_parallel)
					break;
				int flag = 1;
				list<int>::iterator it = P->L.begin();
				for (it; it != P->L.end(); it++) {
					flag = compare_parallel(Block[i], Block[*it]);
					if (flag == 1)
						break;
				}

				if (flag == 0 && P->L.size() < max_parallel) {
					block_pro++;
					P->L.push_back(i);
				}
			}
		}

	}


		while (P->next != NULL) {
			P = P->next;
			if (P->L.size() == max_parallel) {

			}
			if (P->L.size() < max_parallel) {

				for (int i = 0; i < block_num; i++) {
					if (P->L.size() == max_parallel)
						break;
					int flag = 1;
					list<int>::iterator it = P->L.begin();
					for (it; it != P->L.end(); it++) {
						flag = compare_parallel(Block[i], Block[*it]);
						if (flag == 1)
							break;
					}

					if (flag == 0 && P->L.size() < max_parallel) {
						block_pro++;
						P->L.push_back(i);
					}
				}
			}

		}
	
	return head;

}

int find_conflict_x(vector<bool> a, vector<bool> b, int level_a, int level_b) {
	int level = level_a < level_b ? level_a : level_b;

	if (level == 1)
		return 1;//�г�ͻ
	else {
		int block_x;
		for (block_x = 1; block_x < level; block_x++) {
			if (a[block_x] != b[block_x]) {
				return 0;//����ͻ
				break;
			}
		}
		if (block_x == level)
			return 1;
	}
}
void find_conflict(Node_conflict* B_conf, Parallel* P, BS_node* Block, int num_bid_conf) {//�Ҳ��п�֮���ͻ������
	
	int flag_x, flag_y, flag_z;
	Parallel* x_conf, * y_conf, * z_conf;
	Parallel* x_head, * y_head, * z_head;
	//int* id_conf = new int[num];

	int id_i = 0;
	flag_x = flag_y = flag_z = 0;
	x_conf = new Parallel();
	y_conf = new Parallel();
	z_conf = new Parallel();

	x_head = x_conf;
	y_head = y_conf;
	z_head = z_conf;

		list<int>::iterator it = P->L.begin();
		for (it; it != P->L.end(); it++) {
		
			flag_x = find_conflict_x(Block[*it].x_id, Block[num_bid_conf].x_id, Block[*it].level_x, Block[num_bid_conf].level_x);
			flag_y = find_conflict_x(Block[*it].y_id, Block[num_bid_conf].y_id, Block[*it].level_y, Block[num_bid_conf].level_y);
			flag_z = find_conflict_x(Block[*it].z_id, Block[num_bid_conf].z_id, Block[*it].level_z, Block[num_bid_conf].level_z);
		
			if (flag_x == 1) {
				B_conf[*it].coe_x *= 0.5;
				B_conf[num_bid_conf].coe_x *= 0.5;
			}
			if (flag_y == 1) {
				B_conf[*it].coe_y *= 0.5;
 				B_conf[num_bid_conf].coe_y *= 0.5;
			}
			if (flag_z == 1) {
				B_conf[*it].coe_z *= 0.5;
				B_conf[num_bid_conf].coe_z *= 0.5;
			}
	
		}
/*	list<int>::iterator it = P->L.begin();
	for (it; it != P->L.end(); it++) {
		id_conf[id_i] = *it;
		id_i++;
	}

	x_conf->L.push_back(id_conf[0]);
	y_conf->L.push_back(id_conf[0]);
	z_conf->L.push_back(id_conf[0]);
	x_conf->next = NULL;
	y_conf->next = NULL;
	z_conf->next = NULL;

	for (int i = 1; i < num; i++) {

		x_conf = x_head;
		while (x_conf != NULL) {
			flag_x = find_conflict_x(Block[id_conf[i]], Block[x_conf->L.front()], Block[id_conf[i]].level_x, Block[x_conf->L.front()].level_x);

			if (flag_x == 1) {
				x_conf->L.push_front(id_conf[i]);
				break;
			}
			x_conf = x_conf->next;
		}
		if (flag_x == 0) {
			Parallel* s;
			s = new Parallel;
			s->L.push_back(id_conf[i]);
			s->next = NULL;
			//x_conf->next = s;
			x_conf = s;
		}

		y_conf = y_head;
		while (y_conf != NULL) {
			flag_x = find_conflict_x(Block[id_conf[i]], Block[y_conf->L.front()], Block[id_conf[i]].level_y, Block[y_conf->L.front()].level_y);

			if (flag_x == 1) {
				y_conf->L.push_front(id_conf[i]);
				break;
			}
			y_conf = y_conf->next;
		}
		if (flag_x == 0) {
			Parallel* s;
			s = new Parallel;
			s->L.push_back(id_conf[i]);
			s->next = NULL;
			//y_conf->next = s;
			y_conf = s;
		}

		z_conf = z_head;
		while (z_conf != NULL) {
			flag_x = find_conflict_x(Block[id_conf[i]], Block[z_conf->L.front()], Block[id_conf[i]].level_z, Block[z_conf->L.front()].level_z);

			if (flag_x == 1) {
				z_conf->L.push_front(id_conf[i]);
				break;
			}
			z_conf = z_conf->next;
		}
		if (flag_x == 0) {
			Parallel* s;
			s = new Parallel;
			s->L.push_back(id_conf[i]);
			s->next = NULL;
			//z_conf->next = s;
			z_conf = s;
		}

	}

	x_conf = x_head;
	y_conf = y_head;
	z_conf = z_head;
	int num_conf = 1;

	while (x_conf != NULL) {
		num_conf = x_conf->L.size();
		list<int>::iterator it = x_conf->L.begin();
		for (it; it != x_conf->L.end(); it++) {
			B_conf[*it].coe_x = 1 / num_conf;
		}
	}

	while (y_conf != NULL) {
		num_conf = y_conf->L.size();
		list<int>::iterator it = y_conf->L.begin();
		for (it; it != y_conf->L.end(); it++) {
			B_conf[*it].coe_y = 1 / num_conf;
		}
	}

	while (y_conf != NULL) {
		num_conf = z_conf->L.size();
		list<int>::iterator it = z_conf->L.begin();
		for (it; it != z_conf->L.end(); it++) {
			B_conf[*it].coe_z = 1 / num_conf;
		}
	}*/
}

void find_conflict_list(Node_conflict* B_conf, Parallel* P, BS_node* Block) {//�Ҳ��п�֮���ͻ������

	int flag_x, flag_y, flag_z;
	Parallel* x_conf, * y_conf, * z_conf;
	Parallel* x_head, * y_head, * z_head;
	//int* id_conf = new int[num];

	int id_i = 0;
	flag_x = flag_y = flag_z = 0;
	x_conf = new Parallel();
	y_conf = new Parallel();
	z_conf = new Parallel();

	x_head = x_conf;
	y_head = y_conf;
	z_head = z_conf;

	list<int>::iterator it = P->L.begin();
	for (it; it != P->L.end(); it++) {

		list<int>::iterator it_t = it;
		it_t++;
		for (it_t; it_t != P->L.end(); it_t++) {
			flag_x = find_conflict_x(Block[*it].x_id, Block[*it_t].x_id, Block[*it].level_x, Block[*it_t].level_x);
			flag_y = find_conflict_x(Block[*it].y_id, Block[*it_t].y_id, Block[*it].level_y, Block[*it_t].level_y);
			flag_z = find_conflict_x(Block[*it].z_id, Block[*it_t].z_id, Block[*it].level_z, Block[*it_t].level_z);

			if (flag_x == 1) {
				B_conf[*it].coe_x *= 0.5;
				B_conf[*it_t].coe_x *= 0.5;
			}
			if (flag_y == 1) {
				B_conf[*it].coe_y *= 0.5;
				B_conf[*it_t].coe_y *= 0.5;
			}
			if (flag_z == 1) {
				B_conf[*it].coe_z *= 0.5;
				B_conf[*it_t].coe_z *= 0.5;
			}
		
		}

	}
	
}




Parallel* search_parallel_block(BS_node* Block, Node_conflict* B_conf, int block_num, int& num_parallel) {
	num_parallel = 1;
	list<int> size_one;//�洢P��ֻ��һ��block�Ľڵ�
	//int* size_one = new int[];
	int max_parallel = 0;
	Parallel* P, * head, * tmp;
	P = new Parallel();
	head = new Parallel();
	tmp = new Parallel();
	P->next = NULL;
	head = P;

	tmp->L.push_back(0);
	tmp->next = P->next;
	P->next = tmp;
	for (int i = 1; i < block_num; i++) {
		int flag = 0;
		P = head;
		while (P->next != NULL) {
			P = P->next;
			list<int>::iterator it = P->L.begin();
			for (it; it != P->L.end(); it++) {
				flag = compare_parallel(Block[i], Block[*it]);//�ж��Ƿ���Բ���
				if (flag == 1)
					break;
			}
			if (flag == 0) {
				P->L.push_back(i);
				int num_L = P->L.size();
				max_parallel = max_parallel > num_L ? max_parallel : num_L;
				break;
			}
		}
		if (flag == 1) {
			num_parallel++;
			Parallel* s;
			s = new Parallel;
			s->L.push_back(i);
			s->next = NULL;
			P->next = s;
			P = s;
		}
		
	}

	P = head;
	while (P->next != NULL) {
		P = P->next;
		list<int>::iterator it = P->L.begin();
		for (it; it != P->L.end(); it++) {
			B_conf[*it].coe_x = 1;
			B_conf[*it].coe_y = 1;
			B_conf[*it].coe_z = 1;
		}
	}

	//ȡ��P��ֻ��һ����Ľڵ�
	P = head;
	Parallel* front;

	while (P->next != NULL) {
		front = P;
		P = P->next;
		int num = P->L.size();
		if (num == 1) {
			size_one.push_back(P->L.front());
			front->next = P->next;
			P = front;
		}

	}

	//�����������С��max_parallel�Ľڵ�
	P = head;
	int size;
	while (P->next != NULL) {
		P = P->next;
		int num = P->L.size();
		size = size_one.size();
		if (size > 0) {
			for (int i = num; i < max_parallel; i++) {
				//�ҳ�ͻ
				find_conflict(B_conf, P, Block, size_one.front());
				P->L.push_back(size_one.front());
				size_one.pop_front();
				
			}			
		}
		size = size_one.size();
		if (size == 0)
			break;
	}

	if (size > 0) {
		while (P->next != NULL) {
			Parallel* tmp_p = new Parallel();
			tmp_p->L.push_back(size_one.back());
			size_one.pop_back();
			tmp_p->next = NULL;
			P->next = tmp_p;
			list<int>::iterator it = P->L.begin();
			for (it; it != P->L.end(); it++) {
				B_conf[*it].coe_x = 1;
				B_conf[*it].coe_y = 1;
				B_conf[*it].coe_z = 1;
			}
			P = P->next;
		}
	}
	return head;
}

/*
int compare_tree(vector<bool> id, vector<bool> x, int level_tree, int level_block, int block_id) {
	if (level_tree < level_block && block_id == -1) {
		return 1;
	}	
	else {
		for (int block_id = 1; block_id < level_tree; block_id++) {
			if (x[block_id] ^ id[block_id]) {   //���  ��ͬΪ1����ͬΪ0
				return 1;   //����1Ϊ����ͻ,���Բ���
			}
		}
	}
	return 0;
}
*/
int compare_tree(BTnode* BT, BS_node Block) {
	int flag = 0;
	int x_level = BT->x.size() < Block.x_id.size() ? BT->x.size() : Block.x_id.size();
	int y_level = BT->y.size() < Block.y_id.size() ? BT->y.size() : Block.y_id.size();
	int z_level = BT->z.size() < Block.z_id.size() ? BT->z.size() : Block.z_id.size();

	if (BT->block != -1) {
		for (int block_id = 1; block_id < x_level; block_id++) {
			if (BT->x[block_id] ^ Block.x_id[block_id]) {   //���  ��ͬΪ1����ͬΪ0
				flag = 1;   //����1Ϊ����ͻ,���Բ���
				break;
			}
		}
		if (flag == 0)
			return 0;

		flag = 0;
		for (int block_id = 1; block_id < y_level; block_id++) {
			if (BT->y[block_id] ^ Block.y_id[block_id]) {   //���  ��ͬΪ1����ͬΪ0
				flag = 1;   //����1Ϊ����ͻ,���Բ���
				break;
			}
		}
		if (flag == 0)
			return 0;

		flag = 0;
		for (int block_id = 1; block_id < z_level; block_id++) {
			if (BT->z[block_id] ^ Block.z_id[block_id]) {   //���  ��ͬΪ1����ͬΪ0
				flag = 1;   //����1Ϊ����ͻ,���Բ���
				break;
			}
		}
		if (flag == 0)
			return 0;
		else
			return 1;
	}
	else if (BT->block == -1) {
		if (BT->x.size() < Block.x_id.size() && BT->y.size() < Block.y_id.size() && BT->z.size() < Block.z_id.size()) {
			return 1;  //x,y,z�����껹���Լ������±Ƚ�				
		}

		if (BT->x.size() >= Block.x_id.size() ) {    //����ȥif flag=0
			for (int block_id = 1; block_id < Block.x_id.size(); block_id++) {
				if (BT->x[block_id] ^ Block.x_id[block_id]) {   //���  ��ͬΪ1����ͬΪ0
					flag = 1;
					break;
				}
			}
			if (flag == 0) //id��ͻ
				return 0;
		}
		
		flag = 0;
		if (BT->y.size() >= Block.y_id.size()) {
			for (int block_id = 1; block_id < Block.y_id.size(); block_id++) {
				if (BT->y[block_id] ^ Block.y_id[block_id]) {   //���  ��ͬΪ1����ͬΪ0
					flag = 1;
					break;
				}
			}
			if (flag == 0)
				return 0;
		}
		
		flag = 0;
		if (BT->z.size() >= Block.z_id.size()) {
			for (int block_id = 1; block_id < Block.z_id.size(); block_id++) {
				if (BT->z[block_id] ^ Block.z_id[block_id]) {   //���  ��ͬΪ1����ͬΪ0
					flag = 1;
					break;
				}
			}
			if (flag == 0)
				return 0;
		}
		
		if (flag == 1)
			return 1;		
	}
	
	/*
	if (level_tree < level_block && block_id == -1) {
		return 1;
	}
	else {
		for (int block_id = 1; block_id < level_tree; block_id++) {
			if (x[block_id] ^ id[block_id]) {   //���  ��ͬΪ1����ͬΪ0
				return 1;   //����1Ϊ����ͻ,���Բ���
			}
		}
	}
	return 0;*/
}

int tree_conflict(set<int>& block_parallel, BTnode* BT_conflict) {
	if (BT_conflict == NULL)
		return 0;

	if (BT_conflict->block != -1) {
		block_parallel.erase(BT_conflict->block);
	}

	tree_conflict(block_parallel, BT_conflict->lChild);
	tree_conflict(block_parallel, BT_conflict->rChild);

}

//�Ҳ��п飬������
Parallel* search_parallel_block_Tree(BS_node* Block, Node_conflict* B_conf, int block_num, int& num_parallel, BTnode* BT) {
	Parallel* Parallel_list = new Parallel();
	Parallel* Parallel_head = new Parallel();
	Parallel_head = Parallel_list;
	Parallel* tmp = new Parallel();
	tmp->next = NULL;
	Parallel_list->next = tmp;
	Parallel_list = Parallel_list->next;

	int max_parallel = 0;
	num_parallel = 1;
	//vector<bool> parallel_block_id;	
	//Parallel* P, * head;
	//P = new Parallel();
	//head = new Parallel();
	//P->next = NULL;
	//head = P;

	set<int> block_id;
		
	srand(static_cast<unsigned int>(time(nullptr)));
	for (int i = 0; i < block_num; i++) {
		block_id.insert(i);
	}

	while (block_id.size() != 0) {
		set<int> block_parallel;

		auto it = block_id.begin();
		advance(it, rand() % block_id.size());   //���ѡȡblock_id
		int selected = *it;
		block_id.erase(selected);

		Parallel_list->L.push_back(selected); 
		
		BTnode* BT_tmp = new BTnode();
		BTnode* BT_tmp_head = new BTnode();
		BT_tmp = BT;		
		BT_tmp_head = BT_tmp;

		block_parallel = block_id;
		while (block_parallel.size() != 0) {
			queue<BTnode*> q;
			BT_tmp = BT_tmp_head;
			q.push(BT_tmp);
			while (!q.empty()) {   //������ȱ�����,�ҵ����Բ��е�һ��Block
				if (compare_tree(q.front(), Block[selected]) == 0) {
					BTnode* BT_conflict = new BTnode();
					BT_conflict = q.front();
					tree_conflict(block_parallel, BT_conflict);
					compare_tree(q.front(), Block[selected]);
					//q.front()->block=-
					//q.front()->lChild = q.front()->rChild = NULL;
				}
				else {
					if (q.front()->lChild != NULL)
						q.push(q.front()->lChild);
					if (q.front()->rChild != NULL)
						q.push(q.front()->rChild);
				}
				
				q.pop();
				/*
				int h_tmp = q.front()->h % 3;
				if (h_tmp == 2) {
					if (compare_tree(q.front()->x, Block[selected].x_id, q.front()->x.size(), Block[selected].level_x, q.front()->block) == 0) { //��ͻ
						BTnode* BT_conflict = new BTnode();
						BT_conflict = q.front();
						tree_conflict(block_parallel, BT_conflict);
						q.front()->lChild = q.front()->rChild = NULL;
					}
				}
				else if (h_tmp == 0) {
					if (compare_tree(q.front()->y, Block[selected].y_id, q.front()->y.size(), Block[selected].level_y, q.front()->block) == 0) {
						BTnode* BT_conflict = new BTnode();
						BT_conflict = q.front();
						tree_conflict(block_parallel, BT_conflict);
						q.front()->lChild = q.front()->rChild = NULL;
					}
				}
				else {
					if (compare_tree(q.front()->z, Block[selected].z_id, q.front()->z.size(), Block[selected].level_z, q.front()->block) == 0) {
						BTnode* BT_conflict = new BTnode();
						BT_conflict = q.front();
						tree_conflict(block_parallel, BT_conflict);
						q.front()->lChild = q.front()->rChild = NULL;
					}
				}*/
			}

			while (block_parallel.size() != 0) {
				auto it_p = block_parallel.begin();
				advance(it_p, rand() % block_parallel.size());   //���ѡȡblock_id
				selected = *it_p;
				if (Block[selected].block_num != 0) {
					Parallel_list->L.push_back(selected);
					block_parallel.erase(selected);
					block_id.erase(selected);
					break;
				}
				else {
					block_parallel.erase(selected);
					block_id.erase(selected);
				}
			}
			/*
			if (block_parallel.size() != 0) {
				auto it_p = block_parallel.begin();
				advance(it_p, rand() % block_parallel.size());   //���ѡȡblock_id
				selected = *it_p;
				//selected = *block_parallel.begin();
				Parallel_list->L.push_back(selected);
				block_parallel.erase(selected);
				block_id.erase(selected);
			}*/				
		}

		Parallel* Parallel_tmp = new Parallel();
		Parallel_list->next = Parallel_tmp;
		int num_p = Parallel_list->L.size();
		max_parallel = max_parallel > num_p ? max_parallel : num_p;
		Parallel_list = Parallel_list->next;
		num_parallel++;
	}

	Parallel_list = Parallel_head;
	while (Parallel_list->next != NULL) {
		Parallel_list = Parallel_list->next;
		list<int>::iterator it = Parallel_list->L.begin();
		for (it; it != Parallel_list->L.end(); it++) {
			B_conf[*it].coe_x = 1;
			B_conf[*it].coe_y = 1;
			B_conf[*it].coe_z = 1;
		}
	}

	list<int> size_one;
	//ȡ��P��ֻ��һ����Ľڵ�
	Parallel_list = Parallel_head;
	Parallel* front;

	while (Parallel_list->next != NULL) {
		front = Parallel_list;
		Parallel_list = Parallel_list->next;
		int num = Parallel_list->L.size();
		if (num == 1) {
			size_one.push_back(Parallel_list->L.front());
			front->next = Parallel_list->next;
			Parallel_list = front;
		}

	}

	//�����������С��max_parallel�Ľڵ�
	Parallel_list = Parallel_head;
	int size;
	while (Parallel_list->next != NULL) {
		Parallel_list = Parallel_list->next;
		int num = Parallel_list->L.size();
		size = size_one.size();
		if (size > 0) {
			for (int i = num; i < max_parallel; i++) {
				//�ҳ�ͻ
				find_conflict(B_conf, Parallel_list, Block, size_one.front());
				Parallel_list->L.push_back(size_one.front());
				size_one.pop_front();

			}
		}
		size = size_one.size();
		if (size == 0)
			break;
	}

	if (size > 0) {
		while (Parallel_list->next != NULL) {
			Parallel* tmp_p = new Parallel();
			tmp_p->L.push_back(size_one.back());
			size_one.pop_back();
			tmp_p->next = NULL;
			Parallel_list->next = tmp_p;
			list<int>::iterator it = Parallel_list->L.begin();
			for (it; it != Parallel_list->L.end(); it++) {
				B_conf[*it].coe_x = 1;
				B_conf[*it].coe_y = 1;
				B_conf[*it].coe_z = 1;
			}
			Parallel_list = Parallel_list->next;
		}
	}
	/*
	int block_parallel;
	while (block_id.size() != 0) {
		Parallel* Parallel_tmp = new Parallel();

		auto it = block_id.begin();
		advance(it, rand() % block_id.size());   //���ѡȡblock_id
		int selected = *it;
		block_id.erase(selected);
		Parallel_list->L.push_back(selected);

		BTnode* BT_tmp = new BTnode();
		BTnode* BT_tmp_head = new BTnode();
		BT_tmp = BT;          
		BT_tmp_head = BT_tmp;
		int flag_1 = 1;
		int flag_2 = 1;

		BT_tmp = BT_tmp_head;
		while (flag_1 != 0) {  
			q.push(BT_tmp);
			while (!q.empty()) {   //������ȱ�����,�ҵ����Բ��е�һ��Block
				int h_tmp = q.front()->h % 3;
				if (h_tmp == 2) {
					if (compare_tree(q.front()->x, Block[selected].x_id, q.front()->x.size(), Block[selected].level_x) == 0) { //��ͻ
						q.front()->lChild = q.front()->rChild = NULL;
						q.front()->block = -1;   //��ֹ���һ��Ҷ�ӽڵ�Ҳ��ͻ
					}
				}
				else if (h_tmp == 0) {
					if (compare_tree(q.front()->y, Block[selected].y_id, q.front()->y.size(), Block[selected].level_y) == 0) {
						q.front()->lChild = q.front()->rChild = NULL;
						q.front()->block = -1;
					}
				}
				else {
					if (compare_tree(q.front()->z, Block[selected].z_id, q.front()->z.size(), Block[selected].level_z) == 0) {
						q.front()->lChild = q.front()->rChild = NULL;
						q.front()->block = -1;
					}
				}

				if (q.front()->block != -1) {	//���Բ��е�Block
					selected = q.front()->block;					
					Parallel_list->L.push_back(selected);
					block_id.erase(selected);
					flag_2 = 0;
					break;
				}
				
				if (q.front()->lChild != NULL)
					q.push(q.front()->lChild);
				if (q.front()->rChild != NULL)
					q.push(q.front()->rChild);

				q.pop();
			}

			if (q.size() == 0 && flag_2 == 1) {
				flag_1 = 0;
				Parallel* Parallel_tmp = new Parallel();
				Parallel_list->next = Parallel_tmp;
				int num_p = Parallel_list->L.size();
				num_parallel = num_parallel > num_p ? num_parallel : num_p;
				Parallel_tmp = Parallel_tmp->next;
			}
				

			queue<BTnode*> empty;
			swap(q, empty);
		}

		
	}
*/

	return Parallel_head;
}


Parallel* random_Block(BS_node* Block, int num_block, Node_conflict* B_conf, int& num_parallel) {
	Parallel* Parallel_head = new Parallel();
	Parallel* Parallel_list = new Parallel();
	Parallel_head = Parallel_list;
	set<int> block_id;
	num_parallel = 1;
	srand(static_cast<unsigned int>(time(nullptr)));
	for (int i = 0; i < num_block; i++) {
		block_id.insert(i);
	}
	while (block_id.size() != 0) {
		int num = max_num;
		while (num--) {
			if (block_id.size() == 0)
				break;
			auto it = block_id.begin();
			advance(it, rand() % block_id.size());   //���ѡȡblock_id
			int selected = *it;
			block_id.erase(selected);
			Parallel_list->L.push_back(selected);
		}
		num_parallel++;
		list<int>::iterator it = Parallel_list->L.begin();
		for (it; it != Parallel_list->L.end(); it++) {
			B_conf[*it].coe_x = 1;
			B_conf[*it].coe_y = 1;
			B_conf[*it].coe_z = 1;
		}

		find_conflict_list(B_conf, Parallel_list, Block);

		Parallel* p_tmp = new Parallel();
		Parallel_list->next = p_tmp;
		Parallel_list = Parallel_list->next;

	}
	return Parallel_head;
}

//������Ӧ�����귶Χ
b_node Local(BS_node Block, int I, int J, int K) {
	b_node bs;
	int x_start = 0; int x_end = I;
	int y_start = 0; int y_end = J;
	int z_start = 0; int z_end = K;
	for (int i = 1; i < Block.level_x; i++) {
		int n = (x_start + x_end) / 2;
		if (Block.x_id[i] == 0)
			x_end = n + 1;
		else if (Block.x_id[i] == 1)
			x_start = n + 1;
	}
	for (int i = 1; i < Block.level_y; i++) {
		int n = (y_start + y_end) / 2;
		if (Block.y_id[i] == 0)
			y_end = n + 1;
		else if (Block.y_id[i] == 1)
			y_start = n + 1;
	}
	for (int i = 1; i < Block.level_z; i++) {
		int n = (z_start + z_end) / 2;
		if (Block.z_id[i] == 0)
			z_end = n + 1;
		else if (Block.z_id[i] == 1)
			z_start = n + 1;
	}
	bs.x_start = x_start; bs.x_end = x_end;
	bs.y_start = y_start; bs.y_end = y_end;
	bs.z_start = z_start; bs.z_end = z_end;
	return bs;
}
int ToMatrix(Parallel* P, BS_node* Block, b_node* bs, int* num_bs, int I, int J, int K) {
	int count = 0;
	int size = 0;
	int max_parallel = 0;
	while (P->next != NULL) {

		P = P->next;
		int num = P->L.size();
		max_parallel = max_parallel > num ? max_parallel : num;
		num_bs[size] = num;
		list<int>::iterator it = P->L.begin();
		for (it; it != P->L.end(); it++) {
			b_node bs_t = Local(Block[*it], I, J, K);
			bs[count].x_end = bs_t.x_end;
			bs[count].x_start = bs_t.x_start;
			bs[count].y_end = bs_t.y_end;
			bs[count].y_start = bs_t.y_start;
			bs[count].z_end = bs_t.z_end;
			bs[count].z_start = bs_t.z_start;
			count++;
		}
		size++;
	}
	return max_parallel;
}

/*int find_parallel(BS_node Block, Parallel* a, int& max) {

	int num_t = 1;
	Parallel* tmp, * head;
	tmp = new Parallel();
	head = new Parallel();
	a->next = NULL;

	tmp->L.push_back(0);
	tmp->next = NULL;
	a->next = tmp;
	head = a;
	max = 0;

	for (int i = 1; i < Block.block_num; i++) {
		a = head;
		int flag = 0;

		while (a->next != NULL) {
			if (a->L.size() >= thread_size)
				break;

			flag = 0;
			a = a->next;

			int num = a->L.size();
			max = max > num ? max : num;
			list<int>::iterator it = a->L.begin();
			for (it; it != a->L.end(); it++) {
				if (Block.t[i].x == Block.t[*it].x || Block.t[i].y == Block.t[*it].y || Block.t[i].z == Block.t[*it].z) {
					flag = 1;
					break;
				}
			}
			if (flag == 0 && a->L.size() < thread_size) {
				a->L.push_back(i);
				break;
			}
			else {
				flag = 1;
			}

		}
		if (flag == 1) {
			num_t++;
			Parallel* s;
			s = new Parallel();
			s->L.push_back(i);
			s->next = NULL;
			a->next = s;
			a = s;
		}
	}



	a = head;
	while (a->next != NULL) {
		a = a->next;
		if (a->L.size() < thread_size) {

			for(int i=0;i<Block.block_num;i++){

				if (a->L.size() >= thread_size)
					break;
				int flag = 0;
				list<int>::iterator it = a->L.begin();
				for (it; it != a->L.end(); it++) {
					flag = 0;
					if (Block.t[i].x == Block.t[*it].x || Block.t[i].y == Block.t[*it].y || Block.t[i].z == Block.t[*it].z) {
						flag = 1;
						break;
					}
				}
				if (flag == 0 && a->L.size() < thread_size) {
					a->L.push_back(i);
				}
			}
		}
		if (a->L.size() < max) {
			while (a->L.size() < max) {
				a->L.push_back(-1);
			}
		}
	}

	return num_t;
}

int Preproccess_1(Parallel* P, BS_node* Block, double* t, LF_node* pre, int* b_s, t_node* t_s, int* t_block_num, pre_node* tmp, int max_parallel, int num_parallel, int I, int J, int K) {
	Parallel* head_p = new Parallel();
	head_p = P;

	t_block_num[0] = 0;
	int count = 0;
	int count_b = 1;
	int total = 0;
	int sum = 0;


	while (P->next != NULL) {
		P = P->next;

		b_s[count] = P->L.size();

		list<int>::iterator it = P->L.begin();
		for (it; it != P->L.end(); it++) {

			pre_node* tmp_s = new pre_node();
			tmp_s->next = NULL;
			Parallel* a = new Parallel();

			int max_a;
			int num_t = find_parallel(Block[*it], a, max_a);

			t_s[sum].num_b = num_t;
			t_s[sum].num_p = max_a;
			total += num_t * max_a;
			//t_block_p[count] += num_t * max_a;
			tmp_s->t = new LF_node[num_t * max_a];

			t_block_num[count_b] = t_block_num[count_b - 1] + num_t * max_a;
			int count_a = 0;

			while (a->next != NULL) {
				a = a->next;
				list<int>::iterator it_a = a->L.begin();
				for (it_a; it_a != a->L.end(); it_a++) {
					if (*it_a >= 0) {
					tmp_s->t[count_a].x= Block[*it].t[*it_a].x;
					tmp_s->t[count_a].y = Block[*it].t[*it_a].y;
					tmp_s->t[count_a].z = Block[*it].t[*it_a].z;
					tmp_s->t[count_a].rate = t[Block[*it].t[*it_a].x * J * K + Block[*it].t[*it_a].y * K + Block[*it].t[*it_a].z];
					count_a++;
					}
					else {
						tmp_s->t[count_a].x = 0;
						tmp_s->t[count_a].y = 0;
						tmp_s->t[count_a].z = 0;
						tmp_s->t[count_a].rate = 0;
						count_a++;
					}
				}
			}
			sum++;
			count_b++;
			tmp_s->num_t = count_a;
			tmp->next = tmp_s;
			tmp = tmp_s;
		}

		count++;

	}

	return total;
}
void P_pre(pre_node* tmp, LF_node* pre) {
	int sum = 0;

	while (tmp->next != NULL) {
		tmp = tmp->next;
		for (int i = 0; i < tmp->num_t; i++) {
			pre[sum].x = tmp->t[i].x;
			pre[sum].y = tmp->t[i].y;
			pre[sum].z = tmp->t[i].z;
			pre[sum].rate = tmp->t[i].rate;
			sum++;
		}

	}
}
*/
//��һ��Block�пɲ���,����list����
int find_parallel(BS_node Block, pre_node* S, double* t, double coe_x, double coe_y, double coe_z) {
	pre_node* tmp_s = new pre_node();
	Parallel* b = new Parallel();
	Parallel* tmp_b = new Parallel();
	pre_node* head_s = new pre_node();
	head_s = S;
	Parallel* head;
	head = new Parallel();
	head = b;

	tmp_b->L.push_back(0);
	tmp_b->next = NULL;
	b->next = tmp_b;
	if (S->next == NULL) {
		tmp_s->x.push_back(Block.t[0].x);
		tmp_s->y.push_back(Block.t[0].y);
		tmp_s->z.push_back(Block.t[0].z);
		tmp_s->rate.push_back(t[Block.t[0].x * J * K + Block.t[0].y * K + Block.t[0].z]);
		tmp_s->coe_x.push_back(coe_x);
		tmp_s->coe_y.push_back(coe_y);
		tmp_s->coe_z.push_back(coe_z);
		tmp_s->next = NULL;
		S->next = tmp_s;
	}
	else {
		S->next->x.push_back(Block.t[0].x);
		S->next->y.push_back(Block.t[0].y);
		S->next->z.push_back(Block.t[0].z);
		S->next->rate.push_back(t[Block.t[0].x * J * K + Block.t[0].y * K + Block.t[0].z]);
		S->next->coe_x.push_back(coe_x);
		S->next->coe_y.push_back(coe_y);
		S->next->coe_z.push_back(coe_z);
	}


	int num_t = 1;
	int count_a = 0;
	for (int i = 1; i < Block.block_num; i++) {
		b = head;
		S = head_s;
		int flag = 0;

		while (b->next != NULL) {
			if (b->L.size() >= thread_size)//��󲻳����߳���
				break;

			flag = 0;
			b = b->next;
			S = S->next;

			int num = b->L.size();
			list<int>::iterator it = b->L.begin();
			for (it; it != b->L.end(); it++) {//�ҵ���ͬx,y,z��������һ��list��
				if (Block.t[i].x == Block.t[*it].x || Block.t[i].y == Block.t[*it].y || Block.t[i].z == Block.t[*it].z) {
					flag = 1;
					break;
				}
			}
			if (flag == 0 && b->L.size() < thread_size) {
				b->L.push_back(i);

				S->x.push_back(Block.t[i].x);
				S->y.push_back(Block.t[i].y);
				S->z.push_back(Block.t[i].z);
				S->rate.push_back(t[Block.t[i].x * J * K + Block.t[i].y * K + Block.t[i].z]);
				S->coe_x.push_back(coe_x);
				S->coe_y.push_back(coe_y);
				S->coe_z.push_back(coe_z);

				break;
			}
			else {
				flag = 1;
			}

		}
		if (flag == 1) {
			num_t++;
			Parallel* s;
			s = new Parallel();
			s->L.push_back(i);
			s->next = NULL;
			b->next = s;
			b = s;

			if (S->next == NULL) {
				pre_node* tmp = new pre_node();
				tmp->x.push_back(Block.t[i].x);
				tmp->y.push_back(Block.t[i].y);
				tmp->z.push_back(Block.t[i].z);
				tmp->rate.push_back(t[Block.t[i].x * J * K + Block.t[i].y * K + Block.t[i].z]);
				tmp->coe_x.push_back(coe_x);
				tmp->coe_y.push_back(coe_y);
				tmp->coe_z.push_back(coe_z);
				tmp->next = NULL;
				S->next = tmp;
				S = tmp;
			}
			else {
				S = S->next;
				S->x.push_back(Block.t[i].x);
				S->y.push_back(Block.t[i].y);
				S->z.push_back(Block.t[i].z);
				S->rate.push_back(t[Block.t[i].x * J * K + Block.t[i].y * K + Block.t[i].z]);
				S->coe_x.push_back(coe_x);
				S->coe_y.push_back(coe_y);
				S->coe_z.push_back(coe_z);
			}

		}
	}
	Free_list(b);
	return num_t;
}
void Preproccess(Parallel* P, BS_node* Block, Node_conflict* B_conf, LF_node* pre, double* t, list<int>& num_parallel_p) {
	int count = 0;

	while (P->next != NULL) {
		P = P->next;
		int num = P->L.size();

		pre_node* S = new pre_node();
		S->next = NULL;
		pre_node* head_t = new pre_node;
		head_t = S;
		int max = 0;

		list<int>::iterator it = P->L.begin();
		for (it; it != P->L.end(); it++) {//�ҳ�����block�����п��Բ��е�
			S = head_t;

			if (Block[*it].block_num == 0)
				continue;

			double coe_x = B_conf[*it].coe_x;
			double coe_y = B_conf[*it].coe_y;
			double coe_z = B_conf[*it].coe_z;
			int num_t = find_parallel(Block[*it], S, t, coe_x, coe_y, coe_z);
			max = max > num_t ? max : num_t;

		}

		while (S->next != NULL) {//����������һά�����У���¼���Ʋ��е���
			S = S->next;
			num_parallel_p.push_back(S->x.size());

			while (S->x.size() != 0) {
				pre[count].x = S->x.front();
				pre[count].y = S->y.front();
				pre[count].z = S->z.front();
				pre[count].rate = S->rate.front();
				pre[count].coe_x = S->coe_x.front();
				pre[count].coe_y = S->coe_y.front();
				pre[count].coe_z = S->coe_z.front();
				S->x.pop_front();
				S->y.pop_front();
				S->z.pop_front();
				S->rate.pop_front();
				S->coe_x.pop_front();
				S->coe_y.pop_front();
				S->coe_z.pop_front();

				count++;
			}

		}

	}

}


int Preproccess_list(Parallel* P, BS_node* Block, b_node* bs, int* num_bs, int I, int J, int K) {
	int count = 0;
	int size = 0;
	int max_parallel = 0;
	while (P->next != NULL) {

		
		int num = P->L.size();
		max_parallel = max_parallel > num ? max_parallel : num;
		num_bs[size] = num;
		list<int>::iterator it = P->L.begin();
		for (it; it != P->L.end(); it++) {
			b_node bs_t = Local(Block[*it], I, J, K);
			bs[count].x_end = bs_t.x_end;
			bs[count].x_start = bs_t.x_start;
			bs[count].y_end = bs_t.y_end;
			bs[count].y_start = bs_t.y_start;
			bs[count].z_end = bs_t.z_end;
			bs[count].z_start = bs_t.z_start;
			bs[count].id = *it;
			count++;
		}
		size++;
		P = P->next;
	}
	return max_parallel;

}

//lock_freeԤ����
void ToMatrix_LF(Parallel* P, BS_node* Block, double* t, LF_node* LF, int* num_LS, int I, int J, int K) {
	int count = 0;
	int size = 0;
	int max_parallel = 0;
	while (P->next != NULL) {
		P = P->next;
		int num = 0;
		list<int>::iterator it = P->L.begin();
		for (it; it != P->L.end(); it++) {
			for (int i = 0; i < Block[*it].block_num; i++) {
				int x_b, y_b, z_b;
				x_b = Block[*it].t[i].x;
				y_b = Block[*it].t[i].y;
				z_b = Block[*it].t[i].z;
				LF[count].rate = t[x_b * J * K + y_b * K + z_b];
				LF[count].x = x_b;
				LF[count].y = y_b;
				LF[count].z = z_b;
				count++;
				num++;
			}
		}
		num_LS[size] = num;
		size++;
	}
}

//��ʼ��Block
void initial_B(BS_node* Block) {
	
	Block[0].id = 0;
	Block[0].level_x = 1;
	Block[0].level_y = 1;
	Block[0].level_z = 1;
	Block[0].x_id.push_back(0);
	Block[0].y_id.push_back(0);
	Block[0].z_id.push_back(0);
	Block[0].block_num = 0;
	Block[0].t = (T_node*)malloc(block_s * sizeof(T_node));
}


void block_problem(
	double* t_1,
	double* t,
	double* a,
	double* b,
	double* c,
	LF_node* pre,
	int* num_parallel_pre,
	int num_block,
	int num_parallel,
	int max_parallel,
	LF_node* LF,
	int* num_LF,
	int nnz,
	double rate) {

	ofstream ofs_time, ofs_block, ofs_blockdetail, ofs_bnum;
	string address("C:/Users/12625/Desktop/tensor/A_");
	string str;
	stringstream ss;
	ss << rate;
	ss >> str;
	str += "/";
	address += str;
	string add_error, add_time, add_block, add_blockdetail;
	add_error += address;
	add_error += "error.txt";
	add_time += address;
	add_time += "time.txt";
	add_block += address;
	add_block += "block.txt";
	add_blockdetail += address;
	add_blockdetail += "detail.txt";
	T_node* t_block = new T_node[nnz];
	BS_node* Block;
	Parallel* P, * head;
	Node_conflict* B_conf;

	BTnode* BT = new BTnode();
	BTnode* BT_head = new BTnode(); 
	BT_head = BT;

	Block = new BS_node[nnz];    //���ݿ�
	Block->t = new T_node[block_s];

	P = new Parallel();
	head = new Parallel();

	initial_B(Block);
	initial_t(t_block, t, I, J, K, nnz);

	ofs_blockdetail.open(add_blockdetail, ios::out | ios::in | ios::trunc);
	//ofs_time.open(add_time, ios::out | ios::in | ios::trunc);
	double td1 = get_time();
	num_block = tensor_block(Block, t_block, nnz, I, J, K, BT, BT_head);                   //���ݿ黮��
	td1 = get_time() - td1;
	ofs_blockdetail<< "tensor_block=" << num_block << endl;
	//ofs_time << "tensor_block=" << td1 << endl;
	//ofs_time.close();

	BT = BT_head;
	B_conf = new Node_conflict[num_block];
	//head = random_Block(Block, num_block, B_conf, num_parallel);
	head = search_parallel_block_Tree(Block, B_conf, num_block, num_parallel, BT);
	//head = search_parallel_block(Block, B_conf, num_block, num_parallel);
	P = head;

	/*
	int *num_bs = new int[num_parallel];
	b_node* bs = new b_node[nnz];
	if (flag_preproccess == 1) {
		max_parallel = Preproccess_list(P, Block, bs, num_bs, I, J, K);
	}
*/
	pre = new LF_node[nnz];
	list<int> num_parallel_p;
	int num_parallel_t = 0;
	if (flag_preproccess == 1) {
	//	Preproccess(P, Block, B_conf, pre, t, num_parallel_p);
	}
	int sum = 0;
	int num_pre = num_parallel_p.size();
	num_parallel_pre = new int[num_pre];
	list<int>::iterator it = num_parallel_p.begin();
	for (it; it != num_parallel_p.end(); it++) {
		num_parallel_pre[sum] = *it;
		sum++;
	}
	num_parallel_t = num_pre;

	LF = new LF_node[nnz];
	num_LF = new int[num_parallel];
	if (flag_lockfree == 1) {
		ToMatrix_LF(head, Block, t, LF, num_LF, I, J, K);
	}


	//ofs_bnum.open("C:/Users/12625/Desktop/tensor/block_num.txt", ios::out | ios::in | ios::app);
	
	ofs_block.open(add_block, ios::out | ios::in | ios::trunc);
	//ofs_block << num_block << endl;
	ofs_blockdetail << num_block << endl;
//	ofs_bnum << num_block << endl;
	for (int i = 0; i < num_block; i++) {
		ofs_blockdetail << "BID" << Block[i].id << ":";

		for (int num_1 = 0; num_1 < Block[i].level_x; num_1++)
			ofs_blockdetail << Block[i].x_id[num_1];
		ofs_blockdetail << " ";
		for (int num_2 = 0; num_2 < Block[i].level_y; num_2++)
			ofs_blockdetail << Block[i].y_id[num_2];
		ofs_blockdetail << " ";
		for (int num_3 = 0; num_3 < Block[i].level_z; num_3++)
			ofs_blockdetail << Block[i].z_id[num_3];
		ofs_blockdetail << " ";
		ofs_blockdetail << Block[i].block_num << endl;
		ofs_block << Block[i].block_num << endl;
		//	ofs_block << 0 << ":" << Block[i].t[0].x << " " << Block[i].t[0].y << " " << Block[i].t[0].z << "  ";
		//	ofs_block << 1 << ":" << Block[i].t[1].x << " " << Block[i].t[1].y << " " << Block[i].t[1].z << "  ";
	}
	while (P->next != NULL) {
		P = P->next;
		list<int>::iterator it = P->L.begin();
		int n = P->L.size();
		for (it; it != P->L.end(); it++) {
			ofs_blockdetail << Block[*it].id << " ";
		}
		ofs_blockdetail << endl;
	}
	ofs_blockdetail << endl;

	ofs_block.close();
	ofs_blockdetail.close();
	//ofs_bnum.close();

	P = head;
	//delete[]t_block;
	//delete Block;
	Free_list(P);
	//	delete P;
	//	delete head;
	/*	double* a_t = new double[I * r];
		double* b_t = new double[J * r];
		double* c_t = new double[K * r];
		a_t = a;
		b_t = b;
		c_t = c;
	*/
	sgd_train(t_1, t, a, b, c, pre, num_parallel_pre, num_parallel_t, num_block, num_parallel, max_parallel, LF, num_LF, nnz, rate, num_bs, bs, B_conf);
	//	compare_train(rate, a_t, b_t, c_t, num_block, t, t_1,max,min);

}